#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "Basics.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUSparseMatrix.h"
#include "GPUMatrix.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include "GPUMatrixCUDAKernels.cuh"
#include <functional>
#include "CommonMatrix.h"
#include <iostream> // for cout/cerr
#include <assert.h>

typedef unsigned char byte;

#pragma warning(disable : 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning(disable : 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this

#ifdef _WIN32
// thread local storage to access the current stream, initalize to default stream
extern __declspec(thread)
#else
static
#endif
    hipStream_t t_stream;

template <>
const char* CudaErrString<hipsparseStatus_t>(hipsparseStatus_t)
{
    hipDeviceSynchronize();
    return "(see hipsparse.h & look for hipsparseStatus_t or CUSPARSE_STATUS_xxx)";
}

namespace Microsoft { namespace MSR { namespace CNTK {

#pragma region Constructors and Destructor

template <class ElemType>
GPUSPARSE_INDEX_TYPE GPUSparseMatrix<ElemType>::SecondaryIndexValueAt(size_t idx) const
{
    if (idx + m_sliceViewOffset == 0) return 0;
    GPUSPARSE_INDEX_TYPE value;
    CUDA_CALL(hipMemcpy(&value, SecondaryIndexLocation() + idx, sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyDeviceToHost));

    return value;
}

//-------------------------------------------------------------------------
// construction and conversion
//-------------------------------------------------------------------------

template <class ElemType>
void GPUSparseMatrix<ElemType>::ZeroInit(const MatrixFormat matrixFormat, const DEVICEID_TYPE computeDevice)
{
    if (matrixFormat != MatrixFormat::matrixFormatSparseCSC && matrixFormat != MatrixFormat::matrixFormatSparseCSR &&
        matrixFormat != MatrixFormat::matrixFormatSparseBlockCol && matrixFormat != MatrixFormat::matrixFormatSparseBlockRow)
    {
        LogicError("GPUSparseMatrix:  unsupported sparse matrix format");
        // BUGBUG: Then why even define others?
    }
    Base::ZeroInit(matrixFormat, computeDevice);
    UpdateCachedNzCount(0);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(const size_t numRows, const size_t numCols, const size_t numNZ, DEVICEID_TYPE computeDevice, const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/)
{
    ZeroInit(matrixFormat, computeDevice);
    RequireSizeAndAllocate(numRows, numCols, numNZ, true, false);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(DEVICEID_TYPE computeDevice, const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/)
{
    ZeroInit(matrixFormat, computeDevice);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUMatrix<ElemType>& deepCopy, const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/)
{
    ZeroInit(matrixFormat, deepCopy.GetComputeDeviceId());
    if (!deepCopy.IsEmpty())
        SetValue(deepCopy, matrixFormat);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUSparseMatrix<ElemType>& deepCopy)
{
    ZeroInit(deepCopy.GetFormat(), deepCopy.GetComputeDeviceId());
    DeepCopy(deepCopy);
}

// PrepareDevice - Setup the correct cuda context for an operation
// deviceId - the device on which the operation will take place
//            defaults to -1, which means use matrices current device
template <class ElemType>
DEVICEID_TYPE GPUSparseMatrix<ElemType>::PrepareDevice(DEVICEID_TYPE deviceId /*=-1*/) const
{
    // if default value use current compute device
    DEVICEID_TYPE newId = deviceId >= 0 ? deviceId : GetComputeDeviceId();

    Microsoft::MSR::CNTK::PrepareDevice(newId);
    return newId;
}

template <class ElemType>
/*private*/ void GPUSparseMatrix<ElemType>::DeepCopy(const GPUSparseMatrix<ElemType>& deepCopy)
{
    ChangeDeviceTo(deepCopy.GetComputeDeviceId());
    deepCopy.PrepareDevice();

    // If the source is a slice, then this copy is only the content of the slice.
    RequireSizeAndAllocate(deepCopy.GetNumRows(), deepCopy.GetNumCols(), deepCopy.GetNumNZElements(), deepCopy.GetFormat(), true, false);
    m_sliceViewOffset = 0; // reset to zero as we only start copying the indices starting from the offset in the source matrix

    // BUGBUG? I suspect Data() here should be Buffer() for CSC, although Data() is the same because m_sliceViewOffset == 0
    CUDA_CALL(hipMemcpy(Data_IThinkThisShouldBeBuffer(), deepCopy.NzValues(), deepCopy.NzBytes(), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(MajorIndexLocation(), deepCopy.MajorIndexLocationWithSliceViewOffset(), deepCopy.MajorIndexSize(), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(SecondaryIndexLocation(), deepCopy.SecondaryIndexLocation(), deepCopy.SecondaryIndexSize(), hipMemcpyDeviceToDevice));

    // When slicing not from the start, the offset array must be updated.
    if (deepCopy.m_sliceViewOffset > 0)
    {
        int blocksPerGrid = (int) ceil(1.0 * SecondaryIndexCount() / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _shiftColCSCIndexFromSliceViewToAbsolute<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            SecondaryIndexLocation(),
            SecondaryIndexCount(),
            GetNumNZElements());
    }

    UpdateCachedNzCount(deepCopy.NzCount()); // in case of a slice, the sources NZCount already reflects the count of the slice
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const GPUSparseMatrix<ElemType>& deepCopy)
{
    VerifyWritable(__FUNCTION__);

    DeepCopy(deepCopy);
}

// from CPU
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const CPUSparseMatrix<ElemType>& deepCopy)
{
    VerifyWritable(__FUNCTION__);

    SetFormat(deepCopy.GetFormat());
    if (deepCopy.IsEmpty())
    {
        Reset();
        return;
    }

    if (deepCopy.GetFormat() == matrixFormatSparseCSR)
    {
        // BUGBUG: Does this handle slice-view offset correctly? We should only copy parts.
        SetMatrixFromCSRFormat(deepCopy.RowLocation(), deepCopy.ColLocation(), deepCopy.Data(), deepCopy.GetNumElemAllocated(), deepCopy.GetNumRows(), deepCopy.GetNumCols());
    }
    else if (deepCopy.GetFormat() == matrixFormatSparseCSC)
    {
        // BUGBUG: Does this handle slice-view offset correctly? We should only copy parts.
        SetMatrixFromCSCFormat(deepCopy.ColLocation(), deepCopy.RowLocation(), deepCopy.Data(), deepCopy.GetNumElemAllocated(), deepCopy.GetNumRows(), deepCopy.GetNumCols());
    }
    else if (deepCopy.GetFormat() == matrixFormatSparseBlockCol)
    {
        SetMatrixFromSBCFormat(deepCopy.BlockIdsLocation(), deepCopy.Data(), deepCopy.GetBlockSize(), deepCopy.GetNumRows(), deepCopy.GetNumCols());
    }
    else
        NOT_IMPLEMENTED;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::CopyToCPUSparseMatrix(CPUSparseMatrix<ElemType>& cpuSparseMatrix) const
{
    cpuSparseMatrix.VerifyWritable(__FUNCTION__);

    cpuSparseMatrix.SetFormat(GetFormat());
    if (IsEmpty())
    {
        cpuSparseMatrix.Reset();
        return;
    }

    if (this->GetFormat() == matrixFormatSparseCSR)
    {
        // we need to do conversion because CPUSparseMatrix uses size_t for indexes while GPUSparseMatrix uses int
        cpuSparseMatrix.RequireSizeAndAllocate(GetNumRows(), GetNumCols(), GetNumElemAllocated(), true, false);

        PrepareDevice();

        if (sizeof(GPUSPARSE_INDEX_TYPE) == sizeof(CPUSPARSE_INDEX_TYPE))
        {
            // BUGBUG: Should this be RowLocationWithSliceViewOffset()?
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.RowLocation(), RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.ColLocation(), ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* h_CSRRow = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize());
            CUDA_CALL(hipMemcpy(h_CSRRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            ConvertBuffer(cpuSparseMatrix.RowLocation(), h_CSRRow, SecondaryIndexCount());

            GPUSPARSE_INDEX_TYPE* h_Col = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(ColSize());
            CUDA_CALL(hipMemcpy(h_Col, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
            ConvertBuffer(cpuSparseMatrix.ColLocation(), h_Col, MajorIndexCount());
        }

        CUDA_CALL(hipMemcpy(cpuSparseMatrix.Data(), Data_IThinkThisShouldBeBuffer(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));
    }
    else if (this->GetFormat() == matrixFormatSparseCSC)
    {
        // we need to do conversion because CPUSparseMatrix uses size_t for indexes while GPUSparseMatrix uses int
        cpuSparseMatrix.RequireSizeAndAllocate(GetNumRows(), GetNumCols(), GetNumElemAllocated(), true, false);

        PrepareDevice();
        if (sizeof(GPUSPARSE_INDEX_TYPE) == sizeof(CPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.RowLocation(), RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.ColLocation(), ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* h_CSCCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(ColSize());
            CUDA_CALL(hipMemcpy(h_CSCCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
            ConvertBuffer(cpuSparseMatrix.ColLocation(), h_CSCCol, SecondaryIndexCount());

            GPUSPARSE_INDEX_TYPE* h_Row = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize());
            CUDA_CALL(hipMemcpy(h_Row, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            ConvertBuffer(cpuSparseMatrix.RowLocation(), h_Row, MajorIndexCount());
        }

        CUDA_CALL(hipMemcpy(cpuSparseMatrix.Data(), Data(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));
    }
    else if (this->GetFormat() == matrixFormatSparseBlockCol)
    {
        cpuSparseMatrix.RequireSizeAndAllocate(GetNumRows(), GetNumCols(), GetNumNZElements(), true, false);

        PrepareDevice();
        std::vector<GPUSPARSE_INDEX_TYPE> temp(GetBlockSize());
        CUDA_CALL(hipMemcpy(temp.data(), BlockId2ColOrRow(), GetBlockSize() * sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyDeviceToHost));
        for (size_t i = 0; i < temp.size(); ++i)
            cpuSparseMatrix.BlockIdsLocation()[i] = temp[i];

        cpuSparseMatrix.SetBlockSize(GetBlockSize());

        CUDA_CALL(hipMemcpy(cpuSparseMatrix.Data(), Data(), NzBytes(), hipMemcpyDeviceToHost));
    }
    else
        NOT_IMPLEMENTED;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::CopyToDenseMatrix(GPUMatrix<ElemType>& denseMatrix) const
{
    if (IsEmpty())
    {
        denseMatrix.RequireSize(0, 0);
        return;
    }

    PrepareDevice();
    denseMatrix.RequireSize(GetNumRows(), GetNumCols());

    SyncGuard syncGuard;
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSR || GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
        CUSPARSE_CALL(hipsparseSetStream(cusparseHandle, t_stream));

        if (GetFormat() == MatrixFormat::matrixFormatSparseCSR)
        {
            if (sizeof(ElemType) == sizeof(float))
                CUSPARSE_CALL(hipsparseScsr2dense(cusparseHandle, int(GetNumRows()), int(GetNumCols()), descr, (float*)Buffer(), RowLocation(), ColLocation(), (float*)denseMatrix.Data(), int(GetNumRows())));
            else
                CUSPARSE_CALL(hipsparseDcsr2dense(cusparseHandle, int(GetNumRows()), int(GetNumCols()), descr, (double*)Buffer(), RowLocation(), ColLocation(), (double*)denseMatrix.Data(), int(GetNumRows())));
        }
        else
        {
            if (sizeof(ElemType) == sizeof(float))
                CUSPARSE_CALL(hipsparseScsc2dense(cusparseHandle, int(GetNumRows()), int(GetNumCols()), descr, (float*)Buffer(), RowLocation(), ColLocation(), (float*)denseMatrix.Data(), int(GetNumRows())));
            else
                CUSPARSE_CALL(hipsparseDcsc2dense(cusparseHandle, int(GetNumRows()), int(GetNumCols()), descr, (double*)Buffer(), RowLocation(), ColLocation(), (double*)denseMatrix.Data(), int(GetNumRows())));
        }
        CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
    }
    else if (GetFormat() == MatrixFormat::matrixFormatSparseBlockCol || GetFormat() == MatrixFormat::matrixFormatSparseBlockRow)
    {
        denseMatrix.SetValue((ElemType)0);
        ScaleAndAdd(1, *this, denseMatrix);
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

// if the matrix contains strictly one-hot data, then return a vector of the indices; otherwise NULL
template <class ElemType>
size_t* GPUSparseMatrix<ElemType>::TryCopyToArrayAsOneHot() const
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC) // only CSC format for now
        return nullptr;
    let n = GetNumCols();
    if (NzCount() != n) // if not, we know it is not one-hot
        return nullptr;
    // all values must be 1
    vector<ElemType> valBuf(n);
    CUDA_CALL(hipMemcpy(valBuf.data(), Data(), valBuf.size() * sizeof(*valBuf.data()), hipMemcpyDeviceToHost)); // Data() includes slice-view offset
    if (any_of(valBuf.begin(), valBuf.end(), [](ElemType val) { return val != 1; }))
        return nullptr;
    // each column must contain exactly one element
    vector<GPUSPARSE_INDEX_TYPE> secondaryIndexBuf(n+1);
    CUDA_CALL(hipMemcpy(secondaryIndexBuf.data(), SecondaryIndexLocation(), secondaryIndexBuf.size() * sizeof(*secondaryIndexBuf.data()), hipMemcpyDeviceToHost));
    for (size_t j = 0; j < n; j++)
        if (secondaryIndexBuf[j + 1] != secondaryIndexBuf[j] + 1)
            return nullptr;
    // OK! We can get the array now
    vector<GPUSPARSE_INDEX_TYPE> majorIndexBuf(n);
    CUDA_CALL(hipMemcpy(majorIndexBuf.data(), MajorIndexLocationWithSliceViewOffset(), majorIndexBuf.size() * sizeof(*majorIndexBuf.data()), hipMemcpyDeviceToHost)); // note: includes slice-view offset
    unique_ptr<size_t[]> res(new size_t[n]);
    for (size_t j = 0; j < n; j++)
        res[j] = majorIndexBuf[j];
    return res.release();
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ConvertToSparseFormat(MatrixFormat newFormat, GPUSparseMatrix<ElemType>& outMatrix) const
{
    outMatrix.VerifyWritable(__FUNCTION__);

    if (IsEmpty())
    {
        outMatrix.ZeroInit(newFormat, GetComputeDeviceId());
        return;
    }

    MatrixFormat oldFormat = GetFormat();
    if (oldFormat == newFormat)
    {
        outMatrix.SetValue(*this);
        return;
    }

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));

    SyncGuard syncGuard;
    CUSPARSE_CALL(hipsparseSetStream(cusparseHandle, t_stream));

    outMatrix.ChangeDeviceTo(GetComputeDeviceId());
    outMatrix.RequireSizeAndAllocate(GetNumRows(), GetNumCols(), NzCount(), newFormat, true, false);

    if ((oldFormat == matrixFormatSparseCSR && newFormat == matrixFormatSparseCSC) || (oldFormat == matrixFormatSparseCSC && newFormat == matrixFormatSparseCSR))
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            // BUGBUG? I suspect Data() here should be Buffer().
            CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, int(GetNumRows()), int(GetNumCols()), int(GetSizeAllocated()),
                                           (float*) Data_IThinkThisShouldBeBuffer(), RowLocation(), ColLocation(), (float*) outMatrix.Data_IThinkThisShouldBeBuffer(),
                                           outMatrix.RowLocation(), outMatrix.ColLocation(), HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO));
        }
        else
        {
            // BUGBUG? I suspect Data() here should be Buffer().
            CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, int(GetNumRows()), int(GetNumCols()), int(GetSizeAllocated()),
                                           (double*) Data_IThinkThisShouldBeBuffer(), RowLocation(), ColLocation(), (double*) outMatrix.Data_IThinkThisShouldBeBuffer(),
                                           outMatrix.RowLocation(), outMatrix.ColLocation(), HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO));
        }
        InvalidateCachedNzCount();
    }
    else
    {
        NOT_IMPLEMENTED;
    }

    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ConvertToSparseFormat(MatrixFormat newFormat)
{
    if (IsEmpty())
    {
        SetFormat(newFormat);
        return;
    }

    MatrixFormat oldFormat = GetFormat();
    if (oldFormat == newFormat)
        return;

    GPUSparseMatrix<ElemType> tempMatrix(GetComputeDeviceId(), newFormat);
    ConvertToSparseFormat(newFormat, tempMatrix);

    *this = std::move(tempMatrix);
}

template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::CopyToDenseMatrix() const
{
    GPUMatrix<ElemType> res(GetComputeDeviceId());
    if (!IsEmpty())
        CopyToDenseMatrix(res);
    return res;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ChangeDeviceTo(DEVICEID_TYPE to_id)
{
    VerifyWritable(__FUNCTION__);
    if (to_id == CPUDEVICE)
        LogicError("to_id must be valid GPU");
    if (GetComputeDeviceId()== to_id)
        return;

    if (BufferSizeAllocated() == 0) // nothing to move
    {
        assert(Buffer() == nullptr);
    }
    else
    {
        ElemType* d_dst = reinterpret_cast<ElemType*>(TracingGPUMemoryAllocator::Allocate<char>(to_id, BufferSizeAllocated()));

#ifdef WIN32
        // IOMMU DMAR needs to be disabled for CUDA P2P, otherwise it will silently hang.
        // Unfortunately, hipDeviceCanAccessPeer returns true irrespective of the IOMMU settings.
        // More details: https://bugzilla.kernel.org/show_bug.cgi?id=188271
        // http://docs.nvidia.com/cuda/gpudirect-rdma/#supported-systems
        // TODO: enable UVA p2p access once this is fixed.


        // first try peer access
        int canAccessPeer = false;
        CUDA_CALL(hipDeviceCanAccessPeer(&canAccessPeer, to_id, GetComputeDeviceId()));
        if (canAccessPeer)
        {
            hipError_t cudaStatus = hipDeviceEnablePeerAccess(GetComputeDeviceId(), 0);
            if (cudaStatus != hipErrorPeerAccessAlreadyEnabled)
            {
                CUDA_CALL(cudaStatus);
            }
            CUDA_CALL(hipMemcpyPeer(d_dst, to_id, Buffer(), GetComputeDeviceId(), BufferSizeAllocated()));
        }
        else
#endif
        {
            // peer access didn't work, just copy normal
            // make this more efficient by keeping some buffers available for each copy
            ElemType* h_dst = NULL;
            PrepareDevice();
            CUDA_CALL(hipHostMalloc((void**) &h_dst, BufferSizeAllocated()));
            CUDA_CALL(hipMemcpy(h_dst, Buffer(), BufferSizeAllocated(), hipMemcpyDeviceToHost));
            PrepareDevice((DEVICEID_TYPE) to_id);
            CUDA_CALL(hipMemcpy(d_dst, h_dst, BufferSizeAllocated(), hipMemcpyHostToDevice));
            CUDA_CALL(hipHostFree(h_dst));
        }

        TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());
        SetBuffer(d_dst, BufferSizeAllocated());
    }

    SetComputeDeviceId(PrepareDevice(to_id));
}

#if 0
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const CPUMatrix<ElemType>& /*denseMatrix*/)
{
    NOT_IMPLEMENTED;
}
#endif

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& denseMatrix)
{
    VerifyWritable(__FUNCTION__);

    SetValue(denseMatrix, GetFormat());
}

// set value from a dense matrix
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& denseMatrix, const MatrixFormat matrixFormat)
{
    VerifyWritable(__FUNCTION__);

    if (matrixFormat != matrixFormatSparseCSR && matrixFormat != matrixFormatSparseCSC)
    {
        NOT_IMPLEMENTED;
    }

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    int numRows = (int) denseMatrix.GetNumRows(); // m
    int numCols = (int) denseMatrix.GetNumCols(); // n

    int* nnzPerRowOrCol = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(GetComputeDeviceId(), ((matrixFormat & matrixFormatRowMajor) ? numRows : numCols));
    int nnzTotalDevHostPtr = -1;

    {
        SyncGuard syncGuard;
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseSnnz(cusparseHandle, (matrixFormat & matrixFormatRowMajor) ? HIPSPARSE_DIRECTION_ROW : HIPSPARSE_DIRECTION_COLUMN, (int) numRows, (int) numCols, descr,
                                       reinterpret_cast<float*>(denseMatrix.Data()), (int) numRows, nnzPerRowOrCol, &nnzTotalDevHostPtr));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDnnz(cusparseHandle, (matrixFormat & matrixFormatRowMajor) ? HIPSPARSE_DIRECTION_ROW : HIPSPARSE_DIRECTION_COLUMN, (int) numRows, (int) numCols, descr,
                                       reinterpret_cast<double*>(denseMatrix.Data()), (int) numRows, nnzPerRowOrCol, &nnzTotalDevHostPtr));
        }
        // ~SyncGuard
    }

    RequireSizeAndAllocate(numRows, numCols, nnzTotalDevHostPtr, matrixFormat, true, false);

    SyncGuard syncGuard;
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSR)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            // BUGBUG? I suspect Data() here should be Buffer().
            CUSPARSE_CALL(hipsparseSdense2csr(cusparseHandle, (int) GetNumRows(), (int) GetNumCols(), descr, reinterpret_cast<float*>(denseMatrix.Data()),
                                             (int) GetNumRows(), nnzPerRowOrCol, reinterpret_cast<float*>(Data_IThinkThisShouldBeBuffer()), RowLocation(), ColLocation()));
        }
        else
        {
            // BUGBUG? I suspect Data() here should be Buffer().
            CUSPARSE_CALL(hipsparseDdense2csr(cusparseHandle, (int) GetNumRows(), (int) GetNumCols(), descr, reinterpret_cast<double*>(denseMatrix.Data()),
                                             (int) GetNumRows(), nnzPerRowOrCol, reinterpret_cast<double*>(Data_IThinkThisShouldBeBuffer()), RowLocation(), ColLocation()));
        }
    }
    else if (GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            // BUGBUG? I suspect Data() here should be Buffer().
            CUSPARSE_CALL(hipsparseSdense2csc(cusparseHandle, (int) GetNumRows(), (int) GetNumCols(), descr, reinterpret_cast<float*>(denseMatrix.Data()),
                                             (int) GetNumRows(), nnzPerRowOrCol, reinterpret_cast<float*>(Data_IThinkThisShouldBeBuffer()), RowLocation(), ColLocation()));
        }
        else
        {
            // BUGBUG? I suspect Data() here should be Buffer().
            CUSPARSE_CALL(hipsparseDdense2csc(cusparseHandle, (int) GetNumRows(), (int) GetNumCols(), descr, reinterpret_cast<double*>(denseMatrix.Data()),
                                             (int) GetNumRows(), nnzPerRowOrCol, reinterpret_cast<double*>(Data_IThinkThisShouldBeBuffer()), RowLocation(), ColLocation()));
        }
    }
    UpdateCachedNzCount(nnzTotalDevHostPtr);
}

///
/// adjusts the sparse block column matrix with the new Col2BlockId
/// For each column, if new Col2BlockId contains valid index, a corresponding block exists at the index
/// if old col2BlockId[i] contains value at that column, it would be copied over; otherwise the block would be filled with zeros
///
template <class ElemType>
void GPUSparseMatrix<ElemType>::AdjustCol2BlockId(const GPUSPARSE_INDEX_TYPE* cpuCol2BlockId, size_t numBlocks, bool useBlockId2Col)
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseBlockCol)
        LogicError("Expected sparse block col matrix");

    // create new buffer
    size_t numRows = GetNumRows();
    size_t numCols = GetNumCols();
    size_t numNZ = numBlocks * numRows;
    size_t bufferSizeNeeded = BufferSizeNeeded(GetNumRows(), GetNumCols(), numNZ, GetFormat());
    ElemType* pArray = reinterpret_cast<ElemType*>(TracingGPUMemoryAllocator::Allocate<char>(GetComputeDeviceId(), bufferSizeNeeded));
    GPUSPARSE_INDEX_TYPE* newBlockId2Col = (GPUSPARSE_INDEX_TYPE*)(pArray + numNZ);
    GPUSPARSE_INDEX_TYPE* newCol2BlockId = newBlockId2Col + numCols;

    CUDA_CALL(hipMemset(newBlockId2Col, SparseIndex_NotAssigned, numCols * sizeof(GPUSPARSE_INDEX_TYPE)));
    CUDA_CALL(hipMemcpy(newCol2BlockId, cpuCol2BlockId, numCols * sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyHostToDevice));

    int blocksPerGrid = CeilDiv(numCols, GridDim::maxThreadsPerBlock);
 
    // when useBlockId2Col==true, the original col2BlockId is copied to blockId2Col to avoid getting overwritten 
    // during the inplace aggregation of col2BlockId prior to this
    _adjustCol2BlockId<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream >> > (
        numRows,
        numCols,
        useBlockId2Col ? BlockId2ColOrRow() : ColOrRow2BlockId(),
        Data(),
        newCol2BlockId,
        pArray,
        newBlockId2Col);

    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());

    SetBuffer(pArray, bufferSizeNeeded);
    SetSizeAllocated(numNZ);
    SetBlockSize(numBlocks);
}

// fetch the CSC-column/CSR-row offset array from the GPU to the CPU
// Returns a pointer that the caller must delete[].
template <class ElemType>
GPUSPARSE_INDEX_TYPE* GPUSparseMatrix<ElemType>::GetCondensedVector() const
{
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSC || GetFormat() == MatrixFormat::matrixFormatSparseCSR)
    {
        PrepareDevice();
        GPUSPARSE_INDEX_TYPE* pArray = new GPUSPARSE_INDEX_TYPE[SecondaryIndexCount()];
        CUDA_CALL(hipMemcpy(pArray, SecondaryIndexLocation(), sizeof(GPUSPARSE_INDEX_TYPE) * SecondaryIndexCount(), hipMemcpyDeviceToHost));
        return pArray;
    }
    else
    {
        return NULL;
    }
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::MaskColumnsValue(const GPUMatrix<char>& columnsMask, ElemType val, size_t numColsPerMaskEntry)
{
    VerifyWritable(__FUNCTION__);

    if (GetNumCols() != (columnsMask.GetNumCols() * numColsPerMaskEntry))
        RuntimeError("Matrix number of columns must equal 'number of columns in column mask * numColsPerMaskEntry'.");

    if (val != 0)
        LogicError("MaskColumnsValue is not implmented for a non-zero mask for sparse matrices.");

    // We are already done, since the gaps already contain zero by definition.

#ifdef _DEBUG
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        // TODO: We could do this on the GPU, but for now C++ is easier.
        // Download the binary columns mask
        char* maskedCols = columnsMask.CopyToArray();

        // If we're CSC, we only need to verify that the columns to be zeroed are empty, since val == 0.
        // So just download the condensed column vector.
        GPUSPARSE_INDEX_TYPE* colVector = GetCondensedVector();

        // Verify that if the column is to be masked, there are no elements in it.
        size_t n = columnsMask.GetNumCols();
        #pragma omp parallel for
        for (long j = 0; j < n; j++)
            for (long k = 0; k < numColsPerMaskEntry; ++k)
                if (maskedCols[j] == 0 && colVector[(j * numColsPerMaskEntry) + k + 1] != colVector[(j * numColsPerMaskEntry) + k])
                    RuntimeError("GPUSparseMatrix attempted to mask column %d, but it has %d elements in it.", (int)((j * numColsPerMaskEntry) + k), (int)(colVector[(j * numColsPerMaskEntry) + k + 1] - colVector[(j * numColsPerMaskEntry) + k]));
    }
    else
        NOT_IMPLEMENTED;
#endif
}

// assignment is deep
template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(const GPUSparseMatrix<ElemType>& deepCopy)
{
    if (this != &deepCopy)
        SetValue(deepCopy);

    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(GPUSparseMatrix<ElemType>&& moveFrom)
{
    Base::ShallowCopyFrom(moveFrom);
    moveFrom.ZeroValues(); // so that memory in moveFrom is not freed
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(GPUSparseMatrix<ElemType>&& moveFrom)
{
    if (this != &moveFrom)
    {
        Base::ShallowCopyFrom(moveFrom);
        moveFrom.ZeroValues();
    }

    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>::~GPUSparseMatrix()
{
    ZeroValues(); // TODO: why is this necessary?
}

//ResizeAsAndCopyIndexFrom - Resize this sparse matrix to have the same element structure as the passed matrix
// a - sparse matrix whose structure we want to clone
// remark: this was done for element wise operations where the structure will be identical after an operation
template <class ElemType>
void GPUSparseMatrix<ElemType>::ResizeAsAndCopyIndexFrom(const GPUSparseMatrix<ElemType>& a, const bool growOnly /*= true*/)
{
    RequireSizeAndAllocate(a.GetNumRows(), a.GetNumCols(), a.NzCount(), a.GetFormat(), growOnly, false);

    CUDA_CALL(hipMemcpy(MajorIndexLocation(), a.MajorIndexLocation(), MajorIndexSize(), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(SecondaryIndexLocation(), a.SecondaryIndexLocation(), SecondaryIndexSize(), hipMemcpyDeviceToDevice));
    UpdateCachedNzCount(a.NzCount());
}

//-------------------------------------------------------------------------
// main operations
//-------------------------------------------------------------------------

// unlike dense matrices, Reshape() is involved for sparse
template <class ElemType>
void GPUSparseMatrix<ElemType>::Reshape(const size_t numRows, const size_t numCols)
{
    if (GetNumRows() == numRows && GetNumCols() == numCols)
        return;

    VerifyWritable(__FUNCTION__);

    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    if (GetNumRows() * GetNumCols() != numRows * numCols)
        LogicError("GPUSparseMatrix::Reshape: new matrix size does not match current size, can't be reshaped. Did you mean to resize?");

    size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, GetSizeAllocated(), GetFormat());

    ElemType* pArray = reinterpret_cast<ElemType*>(TracingGPUMemoryAllocator::Allocate<char>(GetComputeDeviceId(), bufferSizeNeeded));

    if (Buffer() != nullptr)
    {
        // BUGBUG? I suspect Data() here should be Buffer().
        CUDA_CALL(hipMemcpy(pArray, Data_IThinkThisShouldBeBuffer(), GetSizeElemAllocated(), hipMemcpyDeviceToDevice));

        GPUSPARSE_INDEX_TYPE* majorIndexInNewBuffer = (GPUSPARSE_INDEX_TYPE*) (pArray + GetSizeAllocated());
        GPUSPARSE_INDEX_TYPE* secondaryIndexInNewBuffer = majorIndexInNewBuffer + MajorIndexCount(numRows, numCols, GetSizeAllocated(), GetFormat());

        int blocksPerGrid = (int) ceil(1.0 * numCols / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        // update the indices to represent the reshaping operation (element values remain unchanged)
        _reshape<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            GetNumRows(),                // old row count
            GetNumCols(),                // old col count
            numRows,                  // new row count
            numCols,                  // new col count
            MajorIndexLocation(),     // old row index array
            SecondaryIndexLocation(), // old column index array
            majorIndexInNewBuffer,    // new row index array
            secondaryIndexInNewBuffer // new column index array
            );
        TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());
    }

    SetBuffer(pArray, bufferSizeNeeded);
    SetNumRows(numRows);
    SetNumCols(numCols);
}

// Reserves space for numNZElemToReserve non-zero elements. Also verifies that the matrix is indeed [numRows x numCols].
// If keepExistingValues then the object is assumed already in valid state. This is currently only used for MultiplyAndAdd() for SBC format.
// If not keepExistingValues, the memory is 0-initialized.
template <class ElemType>
void GPUSparseMatrix<ElemType>::Allocate(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const bool growOnly, bool keepExistingValues)
{
    if (GetNumRows() != numRows || GetNumCols() != numCols)
        LogicError("Error, calling allocate with dimensions (%d, %d), but the matrix has dimension (%d, %d).", (int)numRows, (int)numCols, (int)GetNumRows(), (int)GetNumCols());

    size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, numNZElemToReserve, GetFormat());
    bool reallocate = (BufferSizeAllocated() < bufferSizeNeeded || (!growOnly && BufferSizeAllocated() > bufferSizeNeeded));

    if (reallocate)
    {
        // Note that we are allocating one buffer for all of our data structures. Thus the ElemType* nzValues array lives directly next to
        // the GPUSPARSE_INDEX_TYPE* rowIndices/colIndices in sparseCSC/CSR formats. Thus we allocate the number of bytes, and then set the
        // start pointer to an ElemType*.
        char* buf = TracingGPUMemoryAllocator::Allocate<char>(GetComputeDeviceId(), bufferSizeNeeded);
        ElemType* pArray = (ElemType*)(buf);

        // Note this is required due to m_nz 
        CUDA_CALL(hipMemsetAsync(pArray, 0, bufferSizeNeeded, t_stream));
        if (Buffer() != nullptr)
        {
            if (keepExistingValues)
            {
                if (NzCount() > numNZElemToReserve || BufferSizeAllocated() > bufferSizeNeeded)
                    LogicError("Allocate: To keep values, m_nz should <= numNZElemToReserve.");

                // BUGBUG? I suspect Data() here should be Buffer().
                CUDA_CALL(hipMemcpyAsync(pArray, Data_IThinkThisShouldBeBuffer(), GetSizeElemAllocated(), hipMemcpyDeviceToDevice, t_stream));

                GPUSPARSE_INDEX_TYPE* majorIndexInNewBuffer = (GPUSPARSE_INDEX_TYPE*) (pArray + numNZElemToReserve);

                CUDA_CALL(hipMemcpyAsync(majorIndexInNewBuffer, MajorIndexLocation(), MajorIndexSize(), hipMemcpyDeviceToDevice, t_stream));

                GPUSPARSE_INDEX_TYPE* secondaryIndexInNewBuffer = majorIndexInNewBuffer + MajorIndexCount(numRows, numCols, numNZElemToReserve, GetFormat());
                CUDA_CALL(hipMemcpyAsync(secondaryIndexInNewBuffer, SecondaryIndexLocation(), SecondaryIndexSize(), hipMemcpyDeviceToDevice, t_stream));
            }
            TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());
        }

        SetBuffer(pArray, bufferSizeNeeded);
        SetSizeAllocated(numNZElemToReserve);
        if (!keepExistingValues)
            UpdateCachedNzCount(0);
    }
    else
    {
        SetSizeAllocated(ElemCountFromBufferSize(numRows, numCols, GetFormat(), BufferSizeAllocated()));
        // if requested size is smaller, make sure we still initialize to 0 as if it had been reallocated
        if (!keepExistingValues)
            CUDA_CALL(hipMemsetAsync(Buffer(), 0, BufferSizeAllocated(), t_stream));
        UpdateCachedNzCount(0, /*shouldVerify=*/false);
    }
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::RequireSizeAndAllocate(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve /*= 10000*/, const bool growOnly /*= true*/, bool keepExistingValues /*= false*/)
{
    RequireSizeAndAllocate(numRows, numCols, numNZElemToReserve, GetFormat(), growOnly, keepExistingValues);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::RequireSizeAndAllocate(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const MatrixFormat matrixFormat, const bool growOnly, bool keepExistingValues)
{
    RequireSize(numRows, numCols, numNZElemToReserve, matrixFormat, growOnly); // (does nothing if type and numRows/numCols already match, irrespective of numNZElemToReserve)

    if (matrixFormat != GetFormat())
        LogicError("RequireSizeAndAllocate: matrixFormat not set?");

    // this test is redundant; we only short-circuit a comparison of dimensions
    //size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, numNZElemToReserve, matrixFormat);
    //bool reallocate = (BufferSizeAllocated() < bufferSizeNeeded || (!growOnly && BufferSizeAllocated() > bufferSizeNeeded));
    //
    //if (reallocate)
        Allocate(numRows, numCols, numNZElemToReserve, growOnly, keepExistingValues);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::RequireSize(const size_t numRows, const size_t numCols, const bool growOnly /*= true*/)
{
    RequireSize(numRows, numCols, GetFormat(), growOnly);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::RequireSize(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const MatrixFormat matrixFormat, const bool growOnly /*= true*/)
{
    if (GetFormat() != matrixFormat || GetNumRows() != numRows || GetNumCols() != numCols)
        Resize(numRows, numCols, numNZElemToReserve, matrixFormat, growOnly);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve /*= 10000*/, const bool growOnly /*= true*/)
{
    Resize(numRows, numCols, numNZElemToReserve, GetFormat(), growOnly);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const MatrixFormat matrixFormat, const bool growOnly /*= true*/)
{
    VerifyResizable(__FUNCTION__);

    m_sliceViewOffset = 0;
    SetNumRows(numRows);
    SetNumCols(numCols);
    SetNumStorageRows(numRows);
    SetNumStorageCols(numCols);
    SetFormat(matrixFormat);

    // If we really did resize the number of rows/columns, then we changed the number of nz elements allocated. That is, if we used to have a buffer capable of
    // stroring 100 nz elements and 10 columns in CSC format, but we resized to 20 columns, we can no longer store 100 elements, we can only store 95. 
    // Thus we must reset the number of nz elements which can be stored. So let's compute it now.
    size_t newNzElem = ComputeMaxNZElemFromBufferSize(numRows, numCols, BufferSizeAllocated(), matrixFormat);
    SetSizeAllocated(newNzElem);

    size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, numNZElemToReserve, matrixFormat);
    bool reallocate = (BufferSizeAllocated() < bufferSizeNeeded || (!growOnly && BufferSizeAllocated() > bufferSizeNeeded));

    if (reallocate)
        Allocate(numRows, numCols, numNZElemToReserve, growOnly, /*keepExistingValues=*/false);
    else
        ClearNzCount();
}

// Reset matrix to 0.
template <class ElemType>
void GPUSparseMatrix<ElemType>::Reset()
{
    VerifyWritable(__FUNCTION__);

    ClearNzCount();
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ClearNzCount()
{
    // We are now going to reset m_nz to 0. 
    // To reset m_nz to 0, we must do 2 things.
    //    1. We must clear the secondary column index.    --TODO: Why? It should be considered virgin memory when reused!
    //    2. Set the block size to 0.
    // These requirements can be deduced by the NzCount method.
    CUDA_CALL(hipMemsetAsync(Buffer(), 0, BufferSizeAllocated(), t_stream));
    SetBlockSize(0);
    UpdateCachedNzCount(0, /*shouldVerify=*/false);
}

// copy features to GPU
// TODO: This function should be near-identical to SetMatrixFromCSCFormat(), but SetMatrixFromCSCFormat() has been updated. Merge these.
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetMatrixFromCSRFormat(const GPUSPARSE_INDEX_TYPE* h_CSRRow, const GPUSPARSE_INDEX_TYPE* h_Col, const ElemType* h_Val,
                                                       const size_t nz, const size_t numRows, const size_t numCols, const bool IsOnDevice /*= false*/, const DEVICEID_TYPE devId /*= -1*/)
{
    VerifyWritable(__FUNCTION__);

    if (h_CSRRow == nullptr || h_Col == nullptr || h_Val == nullptr)
        LogicError("SetMatrixFromCSRFormat: nullptr passed in.");
    if (!IsOnDevice && nz != h_CSRRow[numRows] - h_CSRRow[0])
        LogicError("SetMatrixFromCSRFormat: wrong nz value passed in.");

    SetComputeDeviceId(PrepareDevice(devId));

    SetFormat(matrixFormatSparseCSR);
    RequireSizeAndAllocate(numRows, numCols, nz, true, false);

    hipMemcpyKind kind = IsOnDevice ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
    // BUGBUG? I suspect Data() here should be Buffer().
    CUDA_CALL(hipMemcpy(Data_IThinkThisShouldBeBuffer(), h_Val, nz * sizeof(ElemType), kind));

    if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
    {
        // ColSize doesn't work since it requires NzCount() to be usable (RowSize doesn't, since it's the fixed, compressed,
        // dimension. Since NzCount is not available (because the sparse indices which is where the NzCount is computed from
        // haven't been copied in yet), we just tell it how many bytes to copy. That is, nz * sizeof(GPUSPARSE_INDEX_TYPE);
        CUDA_CALL(hipMemcpy(RowLocation(), h_CSRRow, RowSize(), kind));
        CUDA_CALL(hipMemcpy(ColLocation(), h_Col, nz * sizeof(GPUSPARSE_INDEX_TYPE), kind));
    }
    else
    {
        GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + nz);
        ConvertBuffer(pCol, h_Col, MajorIndexCount());

        GPUSPARSE_INDEX_TYPE* pRow = pCol + MajorIndexCount();
        ConvertBuffer(pRow, h_CSRRow, nz);

        CUDA_CALL(hipMemcpy(RowLocation(), pRow, RowSize(), kind));
        CUDA_CALL(hipMemcpy(ColLocation(), pCol, nz * sizeof(GPUSPARSE_INDEX_TYPE), kind));
    }
    UpdateCachedNzCount(nz, IsOnDevice); // (when coming from CPU, nz was already validated)
}

// this function will allocate memory while the caller needs to release it
template <class ElemType>
void GPUSparseMatrix<ElemType>::GetMatrixFromCSRFormat(CPUSPARSE_INDEX_TYPE*& h_CSRRow, CPUSPARSE_INDEX_TYPE*& h_Col, ElemType*& h_Val, size_t& numElemAllocated, size_t& nz, size_t& numRows, size_t& numCols) const
{
    VerifyWritable(__FUNCTION__);

    if (h_CSRRow != nullptr || h_Col != nullptr || h_Val != nullptr)
        LogicError("GetMatrixFromCSRFormat: Passed pointers must be nullptr");

    numElemAllocated = GetNumElemAllocated();
    nz = GetNumNZElements();
    numRows = GetNumRows();
    numCols = GetNumCols();

    if (IsEmpty() || nz == 0)
        return;
    else
    {
        h_Val = new ElemType[numElemAllocated];
        h_CSRRow = new CPUSPARSE_INDEX_TYPE[GetNumRows() + 1];
        h_Col = new CPUSPARSE_INDEX_TYPE[nz];

        PrepareDevice();
        // BUGBUG? I suspect Data() here should be Buffer(), and/or slice view offset should be 0.
        CUDA_CALL(hipMemcpy(h_Val, Data_IThinkThisShouldBeBuffer(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));

        if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(h_CSRRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(h_Col, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + ColSize());
            GPUSPARSE_INDEX_TYPE* pRow = pCol + MajorIndexCount();

            CUDA_CALL(hipMemcpy(pRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(pCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));

            ConvertBuffer(h_Col, pCol, MajorIndexCount());
            ConvertBuffer(h_CSRRow, pRow, SecondaryIndexCount());
        }
    }
}

// Set the matrix to the data given by the three arrays, copying the data to the GPU.
// this version is used from the reader
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetMatrixFromCSCFormat(
    const CPUSPARSE_INDEX_TYPE* h_CSCCol, // [0..numCols-1] starting index into h_Row
    const CPUSPARSE_INDEX_TYPE* h_Row,    // [0..nz-1] row of value, order matches h_Val
    const ElemType* h_Val,                // [0..nz-1] values
    const size_t nz, const size_t numRows, const size_t numCols, const bool IsOnDevice /*= false*/, const DEVICEID_TYPE devId /*= -1*/, DataTransferer* transferer /*= nullptr*/)
{
    VerifyWritable(__FUNCTION__);

    if (h_CSCCol == nullptr || h_Row == nullptr || h_Val == nullptr)
        LogicError("SetMatrixFromCSCFormat: nullptr passed in.");
    if (!IsOnDevice && nz != h_CSCCol[numCols] - h_CSCCol[0])
        LogicError("SetMatrixFromCSCFormat: wrong nz value passed in.");
#if 0 // validate input indices
    if (!IsOnDevice)
    {
        for (size_t j = 0; j < numCols; j++)
        {
            if (h_CSCCol[j] < 0 || h_CSCCol[j] > nz)
                LogicError("SetMatrixFromCSCFormat: h_CSCCol[colIndex=%d] beyond nz=%d", (int)j, (int)nz);
            if (j > 0 && h_CSCCol[j] < h_CSCCol[j - 1])
                LogicError("SetMatrixFromCSCFormat: h_CSCCol[] not in ascending order, %d, %d", (int)h_CSCCol[j - 1], (int)h_CSCCol[j]);
        }
        for (size_t k = 0; k < nz; k++)
            if (h_Row[k] < 0 || h_Row[k] >= numRows)
                LogicError("SetMatrixFromCSCFormat: row index of nz element [%d] out of bounds (%d >= %d)", (int)k, (int)h_Row[k], (int)numRows);
    }
#endif

    SetComputeDeviceId(PrepareDevice(devId));
    SetFormat(matrixFormatSparseCSC);
    RequireSizeAndAllocate(numRows, numCols, nz, /*growOnly=*/true, /*keepExistingValues=*/false);

    if (transferer && IsOnDevice)
        RuntimeError("SetMatrixFromCSCFormat: Currently it is not supported to copy data asynchronously from device to device.");
    // m_nz doesn't exist anymore. How are we going to deal with the NzBytes, RowSize, and ColSize? Do it ourselves of course.

    // copy the non-zero elements
    hipMemcpyKind kind = IsOnDevice ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
    if (transferer)
    {
        // TODO: All RequireSizeAndAllocate should be async and use a transferer.
        // Currently there are some memset operations that can be still executing on the default stream,
        // Here we have to wait for them to finish.
        transferer->RecordComputeStreamSyncPoint();
        transferer->WaitForSyncPointOnAssignStreamAsync();
        // BUGBUG? I suspect Data() here should be Buffer(), and/or slice view offset should be 0.
        transferer->CopyCPUToGPUAsync(h_Val, nz, sizeof(ElemType), Data_IThinkThisShouldBeBuffer());
    }
    else
        // BUGBUG? I suspect Data() here should be Buffer(), and/or slice view offset should be 0.
        CUDA_CALL(hipMemcpy(Data_IThinkThisShouldBeBuffer(), h_Val, nz * sizeof(ElemType), kind));

    // copy the index arrays
    if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE)) // note: this is true
    {
        if (transferer)
        {
            transferer->CopyCPUToGPUAsync(h_Row, nz, sizeof(GPUSPARSE_INDEX_TYPE), RowLocation());
            transferer->CopyCPUToGPUAsync(h_CSCCol, numCols + 1, sizeof(GPUSPARSE_INDEX_TYPE), ColLocation());
        }
        else
        {
            CUDA_CALL(hipMemcpy(RowLocation(), h_Row, sizeof(GPUSPARSE_INDEX_TYPE) * nz, kind));
            CUDA_CALL(hipMemcpy(ColLocation(), h_CSCCol, sizeof(GPUSPARSE_INDEX_TYPE) * (numCols + 1), kind));
        }
    }
    else // TODO: is this branch needed, or can it just throw a logic_error?
    {
        size_t allocSize = sizeof(GPUSPARSE_INDEX_TYPE) * nz + sizeof(GPUSPARSE_INDEX_TYPE) * (numCols + 1);
        GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(allocSize);
        GPUSPARSE_INDEX_TYPE* pRow = pCol + nz;

        ConvertBuffer(pCol, h_CSCCol, (numCols+1));
        ConvertBuffer(pRow, h_Row, nz);

        if (transferer)
        {
            transferer->CopyCPUToGPUAsync(pRow, nz, sizeof(GPUSPARSE_INDEX_TYPE), RowLocation());
            transferer->CopyCPUToGPUAsync(pCol, numCols + 1, sizeof(GPUSPARSE_INDEX_TYPE), ColLocation());
        }
        else
        {
            CUDA_CALL(hipMemcpy(RowLocation(), pRow, sizeof(GPUSPARSE_INDEX_TYPE) * nz, kind));
            CUDA_CALL(hipMemcpy(ColLocation(), pCol, sizeof(GPUSPARSE_INDEX_TYPE) * (numCols + 1), kind));
        }
    }

    // TODO: When coming from the CPU, we can check whether the data is one-hot; and pass that to UpdateCachedNZCount() as well.

    UpdateCachedNzCount(nz, IsOnDevice && !transferer); // (when coming from CPU, nz was already validated)
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetMatrixFromSBCFormat(const size_t* blockIds, const ElemType* val, const size_t numBlocks, const size_t numRows, const size_t numCols)
{
    VerifyWritable(__FUNCTION__);

    if (blockIds == nullptr || val == nullptr)
        LogicError("SetMatrixFromSBCFormat: nullptr passed in.");

    SetFormat(matrixFormatSparseBlockCol);
    SetBlockSize(numBlocks);

    if (numBlocks == 0) return; // ====>

    size_t nz = numBlocks * numRows;
    RequireSizeAndAllocate(numRows, numCols, nz, true, false);

    static std::vector<GPUSPARSE_INDEX_TYPE> gpuBlockId2Col(numCols);
    static std::vector<GPUSPARSE_INDEX_TYPE> gpuCol2BlockId(numCols);

    std::fill(gpuBlockId2Col.begin(), gpuBlockId2Col.end(), SparseIndex_NotAssigned);
    std::fill(gpuCol2BlockId.begin(), gpuCol2BlockId.end(), SparseIndex_NotAssigned);

    #pragma omp parallel for
    for (int i = 0; i < numBlocks; ++i)
    {
        gpuBlockId2Col[i] = (GPUSPARSE_INDEX_TYPE)blockIds[i];
        gpuCol2BlockId[blockIds[i]] = i;
    }

    CUDA_CALL(hipMemcpy(Data(), val, nz * sizeof(ElemType), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(BlockId2ColOrRow(), &gpuBlockId2Col[0], numCols * sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(ColOrRow2BlockId(), &gpuCol2BlockId[0], numCols * sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyHostToDevice));
    InvalidateCachedNzCount(); // (for SBC, it is cheap to recover NzCount)
}

// this function will allocate memory while the caller needs to release it
template <class ElemType>
void GPUSparseMatrix<ElemType>::GetMatrixFromCSCFormat(GPUSPARSE_INDEX_TYPE*& h_CSCCol, GPUSPARSE_INDEX_TYPE*& h_Row, ElemType*& h_Val, size_t& numElemAllocated, size_t& nz, size_t& numRows, size_t& numCols) const
{
    if (h_CSCCol != nullptr || h_Row != nullptr || h_Val != nullptr)
        LogicError("GetMatrixFromCSCFormat: Passed pointers must be nullptr");

    numElemAllocated = GetNumElemAllocated();
    nz = GetNumNZElements();
    numRows = GetNumRows();
    numCols = GetNumCols();

    if (IsEmpty())
        return;
    else
    {
        h_Val = new ElemType[numElemAllocated];
        h_CSCCol = new GPUSPARSE_INDEX_TYPE[GetNumRows() + 1];
        h_Row = new GPUSPARSE_INDEX_TYPE[nz];

        PrepareDevice();
        // BUGBUG? I suspect Data() here should be Buffer(), and/or slice view offset should be 0.
        CUDA_CALL(hipMemcpy(h_Val, Data_IThinkThisShouldBeBuffer(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));

        if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(h_Row, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(h_CSCCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + ColSize());
            GPUSPARSE_INDEX_TYPE* pRow = pCol + SecondaryIndexCount();

            CUDA_CALL(hipMemcpy(pRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(pCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));

            ConvertBuffer(h_CSCCol, pCol, SecondaryIndexCount());
            ConvertBuffer(h_Row, pRow, MajorIndexCount());
        }
    }
}

#pragma endregion Constructors and Destructor

#pragma region Static BLAS Functions

// dense X sparse = dense
// This is e.g. used for the forward pass of an embedding (e = E w where w is one-hot).
template <class ElemType>
void GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA,
                                                       const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, ElemType beta, GPUMatrix<ElemType>& c)
{
    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId() || (lhs.GetComputeDeviceId() != c.GetComputeDeviceId()))
        RuntimeError("GPUSparseMatrix::MultiplyAndWeightedAdd: All matrices must be on the same GPU");

    // BUGBUG: Below we fail if one of the factors is empty.That is wrong. We should be able to handle empty factors (e.g. worker of a minibatch got 0 samples).
    // Probably one should test further down and exit early, but we need to make sure that c is correct for beta != 0.
    if (lhs.IsEmpty() || rhs.IsEmpty())
        LogicError("GPUSparseMatrix::MultiplyAndWeightedAdd:  one of the input matrix is empty.");

    int m = transposeA ? (int) lhs.GetNumCols() : (int) lhs.GetNumRows();
    int k = transposeA ? (int) lhs.GetNumRows() : (int) lhs.GetNumCols();
    int l = transposeB ? (int) rhs.GetNumCols() : (int) rhs.GetNumRows();
    int n = transposeB ? (int) rhs.GetNumRows() : (int) rhs.GetNumCols();

    assert(m > 0 && k > 0 && l > 0 && n > 0); // converting from size_t to int may cause overflow
    assert(k == l);
    if (k != l)
    {
        InvalidArgument("GPUSparseMatrix::MultiplyAndWeightedAdd: The inner dimensions of a (= %d) and b (= %d) don't match.", k, l);
    }

    if (beta == 0)
        c.RequireSize(m, n);
    else
        c.VerifySize(m, n); // Can't resize if beta != 0

    c.PrepareDevice();
    if (rhs.GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        // this is the code branch for embedding from sparse input
        ConvolveAndWeightedAdd(alpha, lhs, transposeA, rhs, transposeB, beta, c, 1, 1, false, false);
    }
    else if (rhs.GetFormat() == matrixFormatSparseCSR)
    {
        GPUSparseMatrix<ElemType> tempMatrix(rhs.GetComputeDeviceId(), matrixFormatSparseCSC);
        rhs.ConvertToSparseFormat(matrixFormatSparseCSC, tempMatrix);
        MultiplyAndWeightedAdd(alpha, lhs, transposeA, tempMatrix, transposeB, beta, c);
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

// dense X sparse = dense
// This is called from MultiplyAndWeightedAdd() for the forward pass of an embedding (e = E w where w is one-hot), with numChannels=1, no subsampling, no padding, not channelwise.
template <class ElemType>
void GPUSparseMatrix<ElemType>::ConvolveAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA,
                                                       const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, ElemType beta,
                                                       GPUMatrix<ElemType>& c, size_t numChannels, size_t horizontalSubsample, bool padding, bool channelwise)
{
    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId() || (lhs.GetComputeDeviceId() != c.GetComputeDeviceId()))
        RuntimeError("GPUSparseMatrix<ElemType>::ConvolveAndWeightedAdd: All matrices must be on the same GPU");

    if (lhs.IsEmpty() || rhs.IsEmpty())
        LogicError("GPUSparseMatrix<ElemType>::ConvolveAndWeightedAdd:  one of the input matrix is empty.");

    int m = transposeA ? (int) lhs.GetNumCols() : (int) lhs.GetNumRows();
    int k = transposeA ? (int) lhs.GetNumRows() : (int) lhs.GetNumCols();
    int l = transposeB ? (int) rhs.GetNumCols() : (int) rhs.GetNumRows();
    int n = transposeB ? (int) rhs.GetNumRows() : (int) rhs.GetNumCols();

    assert(m > 0 && k > 0 && l > 0 && n > 0); // converting from size_t to int may cause overflow

    int numSteps = 0;
    if (padding)
        numSteps = (int) ceil(1.0 * l / (horizontalSubsample * numChannels));
    else if (l >= k)
        numSteps = 1 + (l - k) / (horizontalSubsample * numChannels);

    if (numSteps == 0)
        LogicError("ConvolveAndWeightedAdd: number of steps is zero. Matrix dimensions are incorrect or set padding to true.");

    int cRows = m * numSteps;
    int cCols = n;

    if (beta == 0)
        c.RequireSize(cRows, cCols);
    else
        c.VerifySize(cRows, cCols); // Can't resize if beta != 0

    c.PrepareDevice();
    if (rhs.GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        if (!transposeB)
        {
            int blocksPerGrid = (int) ceil(1.0 * cRows * cCols / GridDim::maxThreadsPerBlock);
            SyncGuard syncGuard;
            _dense1DConvMultSparseCSCAndWeightedAddToDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                m,                   // rowDense
                k,                   // colDense
                n,                   // colSparse
                numChannels,         // number of input channels
                numSteps,            // convolution num steps
                horizontalSubsample, // convolution step size
                channelwise,         // channelwise or pixelwise multiplication
                alpha,
                reinterpret_cast<const ElemType*>(lhs.Data()), // dense
                transposeA,
                reinterpret_cast<const ElemType*>(rhs.Buffer()), // sparse nz values. Note that because of the offsets we use the array
                rhs.RowLocation(),
                rhs.ColLocation(),
                beta,
                reinterpret_cast<ElemType*>(c.Data()) // dense target
                );
        }
        else
        {
            if (beta == 0.0)
                c.SetValue((ElemType)0);
            else if (beta != 1.0)
                RuntimeError("Only support c += alpha * a operation");

            int blocksPerGrid = (int) ceil(1.0 * cRows / GridDim::maxThreadsPerBlock);
            SyncGuard syncGuard;
            for (int rowInB = 0; rowInB < l; rowInB++)
            {
                _dense1DConvMultSparseCSCTransposeAndAddToDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                    m,                   // rowDense
                    k,                   // colDense
                    n,                   // colSparse
                    numChannels,         // number of input channels
                    numSteps,            // convolution num steps
                    horizontalSubsample, // convolution step size
                    channelwise,         // channelwise or pixelwise multiplication
                    rowInB,
                    alpha,
                    reinterpret_cast<const ElemType*>(lhs.Data()), // dense
                    transposeA,
                    reinterpret_cast<const ElemType*>(rhs.Buffer()), // sparse nz values
                    rhs.RowLocation(),
                    rhs.ColLocation(),
                    reinterpret_cast<ElemType*>(c.Data()) // dense target
                    );
            }
        }
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

// c[:,j] = alpha * v[j] * a[:,j] + beta * c[:,j]
// -> dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::ColumnwiseScaleAndWeightedAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& v, ElemType beta, GPUMatrix<ElemType>& c)
{
    if (v.GetNumRows() != 1 && v.GetNumCols() != 1)
        InvalidArgument("the argument v must be a vector"); // v is a vector

    if (a.GetFormat() != matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    if (beta == 0)
    {
        c.RequireSize(a.GetNumRows(), a.GetNumCols());
        c.SetValue((ElemType)0);
    }
    else
        c.VerifySize(a.GetNumRows(), a.GetNumCols()); // Can't resize if beta != 0

    int blocksPerGrid = (int)ceil(1.0 * a.GetNumCols() / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _columnwiseScaleAndWeightedAdd4CSC<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
        alpha,
        a.Buffer(), a.ColLocation(), a.RowLocation(),
        v.Data(),
        beta,
        c.Data(),
        a.GetNumRows(), a.GetNumCols());
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::TensorShuffleScaleAndAdd(ElemType keepWeight, const GPUSparseMatrix<ElemType>& a, size_t D, size_t S, size_t M, size_t K, size_t T, 
    ElemType scaleFactor, const GPUSparseMatrix<ElemType>& b, GPUSparseMatrix<ElemType>& c)
{
    c.VerifyWritable(__FUNCTION__);

    if (a.GetComputeDeviceId() != c.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId())
        RuntimeError("GPUSparseMatrix<ElemType>::TensorShuffleScaleAndAdd: All matrices must be on the same GPU");

    if (a.GetFormat() != MatrixFormat::matrixFormatSparseCSC || b.GetFormat() != MatrixFormat::matrixFormatSparseCSC || c.GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    // Can't distribute the operations if we need to move values across columns
    if (a.GetNumCols() != T || keepWeight != 0 || scaleFactor != 1)
        NOT_IMPLEMENTED;

    if (a.GetNumRows() != D * S * M * K)
        LogicError("GPUSparseMatrix<ElemType>::TensorShuffleScaleAndAdd: tensor dimensions and underlying matrix dimensions don't match");

    c.RequireSizeAndAllocate(a.GetNumRows(), a.GetNumCols(), a.NzCount(), true, false);

    if (a.NzCount() > 0)
    {
        c.PrepareDevice();
        SyncGuard syncGuard;
        CUDA_LONG N = (CUDA_LONG) a.NzCount();
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        _tensorShuffleScaleAndAddRowSparse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            reinterpret_cast<const ElemType*>(a.Buffer()), // source nz values
            a.RowLocation(),
            a.ColLocation(),
            reinterpret_cast<ElemType*>(c.Buffer()), // target nz values
            c.RowLocation(),
            c.ColLocation(),
            D, S, M, K, T,
            a.NzCount());
        c.UpdateCachedNzCount(a.NzCount());
    }
    else
    {
        CUDA_CALL(hipMemset(c.Buffer(), 0, c.BufferSizeAllocated()));
        c.UpdateCachedNzCount(0);
    }
}

// dense X sparse = sparse
// This is the backward pass from hidden layer to feature weight.
// E.g. e = E * w -> grad_E = grad_e * w'   (where w = CSC one-hot).
// In the one-hot case, this adds grad_e(t) to column w_index(t) of E.
template <class ElemType>
/*static*/ void GPUSparseMatrix<ElemType>::MultiplyAndAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA,
                                                          const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, GPUSparseMatrix<ElemType>& c)
{
    c.VerifyWritable(__FUNCTION__);

    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId())
        RuntimeError("GPUSparseMatrix::MultiplyAndAdd: All matrices must be on the same GPU");

    int m = transposeA ? (int) lhs.GetNumCols() : (int) lhs.GetNumRows(); // output dimension
    int k = transposeA ? (int) lhs.GetNumRows() : (int) lhs.GetNumCols(); // inner dimension (sparse gradient: == number of samples)
    int l = transposeB ? (int) rhs.GetNumCols() : (int) rhs.GetNumRows(); // inner dimension (== k required)
    int n = transposeB ? (int) rhs.GetNumRows() : (int) rhs.GetNumCols(); // input dimension

    assert(m > 0 && k > 0 && l > 0 && n > 0);
    if (k != l)
    {
        InvalidArgument("GPUSparseMatrix::MultiplyAndAdd: The inner dimensions of a (= %d) and b (= %d) don't match.", k, l);
    }

    if (!transposeA && !transposeB)
    {
        NOT_IMPLEMENTED;
    }
    else if (!transposeA && transposeB)
    {
        // This is the backward pass from hidden layer to feature weight.
        if (rhs.GetFormat() != matrixFormatSparseCSC)
            NOT_IMPLEMENTED;

        c.SetFormat(matrixFormatSparseBlockCol);

        lhs.PrepareDevice();

        int blocksPerGrid = 0;
        SyncGuard syncGuard;

        // based on the size of m_nz in rhs and numCols in the resulted matrix we use different approaches
        size_t rhs_nz = rhs.NzCount();

        // Block col storage format (target matrix):
        //  - GetBlockSize()               :                  number of non-zero columns
        //  - ColOrRow2BlockId()[colIndex] : [numCols]        storage index (=index into the compact matrix), or SparseIndex_Pending if not determined yet, or SparseIndex_NotAssigned if empty
        //  - BlockId2ColOrRow()[blockId]  : [GetBlockSize()] column index (=logical index into the matrix that this object represents)
        //                                                     This array is allocated as numCols, but only elements up to GetBlockSize()-1 are used.
        // The storage indices can be in any order (they are not sorted).
        size_t blockSizePrev = c.GetBlockSize(); // number of non-zero columns in target matrix. Compact storage contains this many columns.
        if (blockSizePrev == 0)
        {
            //fprintf(stderr, "MultiplyAndAdd: resetting to %d items\n", (int)n), fflush(stderr);
            // the first time, we allocate space for all entries
            // Initially, all columns are empty. As we keep adding matrix products into it, columns
            // flip from empty to non-empty (but never back to empty).
            // This resetting is done lazily. Reset() just resets the block size, and this code here picks up on it and finishes the initialization.
            // Note that this may be expensive, as we initialize the full dimension (which is large, otherwise we wouldn't be using sparse).
            // We could speed that up by maintaining a dirty range, and only resetting that. Reset() could create a "lazy reset" instruction.
            c.Resize(m, n, 0);
            // Note a small hack: hipMemset() sets bytes, but we initialize 32-bit ints. Hence, all bytes in SparseIndex_NotAssigned must be identical (0xff).
            static_assert(SparseIndex_NotAssigned == -1, "SparseIndex_NotAssigned must be 0xffffffff");
            if (n > c.GetNumCols())
                LogicError("MultiplyAndAdd: wrong allocation (primary and secondary indices)?");
            CUDA_CALL(hipMemsetAsync(c.ColOrRow2BlockId(), 0xff, sizeof(GPUSPARSE_INDEX_TYPE) * n, t_stream));
            // PERF BUGBUG: BlockId2ColOrRow()[*] does not need to be initialized actually, does it?
            CUDA_CALL(hipMemsetAsync(c.BlockId2ColOrRow(), 0xff, sizeof(GPUSPARSE_INDEX_TYPE) * n, t_stream));
        }

        // temp buffer to transfer a single value
        size_t* pBlockSizeTempGpu = TracingGPUMemoryAllocator::Allocate<size_t>(lhs.GetComputeDeviceId(), 1);
        // (perf note: we could use a kernel to set the value, to avoid the GPU sync; but below we copy it back, which cannot be avoided)
        CUDA_CALL(hipMemcpyAsync(pBlockSizeTempGpu, &blockSizePrev, sizeof(size_t), hipMemcpyHostToDevice, t_stream));
        // TODO: Can we avoid the memory allocation here?? Just keep around a bunch of general-use buffers?

        // determine which columns are non-zero -> ColOrRow2BlockId()[colIndex]
        // Some columns may already be non-zero. Those already have a storage index.
        // Columns that were zero before but are no longer get SparseIndex_Pending.
        // This is driven by rhs.RowLocation(); that is, the array of row indices of non-zero elements.
        blocksPerGrid = (int) ceil(((double) rhs_nz) / GridDim::maxThreadsPerBlock);
        _findColsWithValues<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            /*in*/rhs.RowLocation(), /*in ref*/rhs.ColLocation()[0], /*out*/c.ColOrRow2BlockId(), rhs_nz);
        // RowLocation = base of nz row-index array, without potential slice-view offset. Kernel offsets it by ColLocation()[0], which is non-zero in case of a slice view.
        // Now ColOrRow2BlockId()[colIndex] contains an index or SparseIndex_Pending for all non-empty columns.

        // assign a storage index to any newly added columns
        blocksPerGrid = (int) ceil(((double) n) / GridDim::maxThreadsPerBlock);
        _determineBlockIds<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            c.BlockId2ColOrRow(), c.ColOrRow2BlockId(), n, pBlockSizeTempGpu);
        // Now all SparseIndex_Pending values in ColOrRow2BlockId()[colIndex] have been replaced,
        // and BlockId2ColOrRow()[storageIndex] values for those have been placed.
        // *pBlockSizeTempGpu has been incremented accordingly.
        // BlockId2ColOrRow()[storageIndex] is now valid up to [*pBlockSizeTempGpu-1].
        // Newly added columns at this point contain a storage index that is out of bounds w.r.t. the compact storage.

        // Retrieve the updated #non-zero columns (*pBlockSizeTempGpu).
        // setting the block size incurs a GPU sync
        // Note: In the case of one-hot, we know an upper bound. We could leverage that to avoid the round-trip/GPU sync.
        // TODO: We could count the number of non-zero rows when transferring from the CPU.
        //       Should we just keep the CPU-side data around? In the one-hot case? Then we can do the mapping CPU-side.
        //       We can then even keep a CPU-side buffer in the weight matrix, for this purpose.
        // Or:
        //       We could also operate with an upper bound, which gets updated asynchronously (just fire off the async copy).
        //       We would then allocate w.r.t. the upper bound (=current + #new samples). At some point in time, the true,
        //       smaller, value would arrive asynchronously. With proper state tracking, we could avoid to unnecessarily
        //       initialize newly aded zero-columns, beyond the upper bound.
        size_t blockSizeCurr;
        CUDA_CALL(hipMemcpy(&blockSizeCurr, pBlockSizeTempGpu, sizeof(size_t), hipMemcpyDeviceToHost));
        TracingGPUMemoryAllocator::Free<size_t>(lhs.GetComputeDeviceId(), pBlockSizeTempGpu);
        c.SetBlockSize(blockSizeCurr);
        // Now GetBlockSize(), ColOrRow2BlockId()[*], and BlockId2ColOrRow()[*] are up to date.
        if (blockSizeCurr > c.GetNumCols())
            LogicError("MultiplyAndAdd: wrong allocation (block size)?");

        // if new storage columns have been added, zero them out (after growing the compact storage if needed)
        if (blockSizeCurr < blockSizePrev)
            LogicError("MultiplyAndAdd: #non-zero columns became less??");
        if (blockSizeCurr > blockSizePrev)
        {
            //fprintf(stderr, "MultiplyAndAdd: growing #non-zero columns from %d to %d, for %d items\n", (int)blockSizePrev, (int)blockSizeCurr, (int)k), fflush(stderr);
            // zero-initialize new blocks that were just added to block storage
            size_t nnz = m * blockSizeCurr;
            c.RequireSizeAndAllocate(m, n, nnz, true, true); // we need to keep the col2blockid and blockid2col info when resizing.
            CUDA_CALL(hipMemsetAsync(c.Buffer() + m * blockSizePrev, 0, sizeof(ElemType) * m * (blockSizeCurr - blockSizePrev), t_stream));
        }
        // Now allocation is up-to-date as well.

        // now perform the actual matrix product, adding into the compact storage
        // This only processes the non-zero columns, which are already determined and passed in via ColOrRow2BlockId().
        LONG64 N = (LONG64) lhs.GetNumElements(); // =m*l, here we process for each row in lhs and each column in rhs (==columns in lhs)
        blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock); // elements of lhs linearly distributed over cores
        if (c.m_sliceViewOffset != 0)
            InvalidArgument("MultiplyAndAdd: Sparse block column matrices cannot be sliced.");
        _denseMulSparseCSCTransposeToSparseBlockCol2<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            alpha,
            // lhs (in)
            /*lhsValues=*/    lhs.Data(), // this is dense
            /*numRowsLhs=*/   m,          // output dimension, height of lhs
            /*numColsRhs=*/   l,          // inner dimension. In the case of the sparse gradient, this is the number of samples.
            // rhs (in)
            /*rhsNZValues=*/  rhs.Buffer(),      // [nzIndex] rhs nz-element array base, without potential slice-view offset.
            /*rhsRows=*/      rhs.RowLocation(), // [nzIndex] rhs index array base, without potential slice-view offset.
            /*rhsCols=*/      rhs.ColLocation(), // [colIndex] first nzIndex for a given column, with potential slice-view offset. End nzIndex is that of the next column.
            // result (out)
            /*col2blockIds=*/ c.ColOrRow2BlockId(), // (in) [colIndex] storage index for each non-zero column
            /*resultValues=*/ c.Buffer());          // (in/out) [rowIndex, storageIndex] pointer to compact storage

        c.InvalidateCachedNzCount(); // (the cached nzCount value is not used for block-sparse; nzCount = GetBlockSize() * numRows)
    }
    else if (transposeA && !transposeB)
    {
        NOT_IMPLEMENTED;
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

// find the rows of rhs with values
template <class ElemType>
size_t GPUSparseMatrix<ElemType>::IdentifyRowsWithValues() const
{
    if (GetFormat() != matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    let nnz = NzCount();
    this->ReserveTempDeviceBuffer(nnz);
    map<size_t, GPUSPARSE_INDEX_TYPE> indexer;
    GPUSPARSE_INDEX_TYPE* rowToId = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(sizeof(GPUSPARSE_INDEX_TYPE) * nnz * 2);

    // In the first nnz values of the 'rowToId' we will store the block ids of the nonzero-values (to be computed below).
    // In the next nnz values of 'rowToId' we store the row-ids of the non-zero values (copied from GPU).
    GPUSPARSE_INDEX_TYPE* h_Row = rowToId + nnz;
    CUDA_CALL(hipMemcpy(h_Row, RowLocation(), sizeof(GPUSPARSE_INDEX_TYPE) * nnz, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < nnz; i++)
    {
        size_t row = h_Row[i];
        if (indexer.find(row) == indexer.end())
        {
            size_t id = indexer.size(); // We need to assign size to a temp variable due to difference in Linux and Windows
            indexer[row] = id;
        }
        rowToId[i] = indexer[row];
    }
    CUDA_CALL(hipMemcpy(GetTempDeviceBuffer(), rowToId, sizeof(GPUSPARSE_INDEX_TYPE) * nnz, hipMemcpyHostToDevice));
    return indexer.size();
}

// -> dense
// used for gradients udpate
template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(const ElemType alpha, const GPUSparseMatrix<ElemType>& lhs, GPUMatrix<ElemType>& rhs)
{
    if (lhs.GetNumRows() != rhs.GetNumRows() || lhs.GetNumCols() != rhs.GetNumCols())
        LogicError("ScaleAndAdd: dimension mismatch");

    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId())
        RuntimeError("GPUSparseMatrix::ScaleAndAdd: All matrices must be on the same GPU");

    if (lhs.GetFormat() == matrixFormatSparseBlockCol || lhs.GetFormat() == matrixFormatSparseBlockRow)
    {
        bool blockCol = (lhs.GetFormat() == matrixFormatSparseBlockCol);

        SyncGuard syncGuard;
        LONG64 N = (LONG64) lhs.GetNumNZElements();
        int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
        _scaleSparseBlockAndAddToDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(
            alpha,
            blockCol,
            lhs.GetNumRows(),
            lhs.GetNumCols(),
            lhs.GetBlockSize(),
            lhs.Data(),
            lhs.BlockId2ColOrRow(),
            rhs.Data());
    }
    else
    {
        ScaleAndAdd(alpha, lhs, 1, rhs, rhs);
    }
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncate(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();

    CUDA_LONG blocksPerGrid = (CUDA_LONG) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    ElemType* values = NzValues();
    _inplaceTruncate<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(values, threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSoftThreshold(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();

    CUDA_LONG blocksPerGrid = (CUDA_LONG) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    ElemType* values = NzValues();
    _inplaceSoftThreshold<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(values, threshold, N);
    return *this;
}

// A helper method used in MomentumSGDUpdate and NesterovAcceleratedMomentumSGDUpdate.
// Modifies the smoothed gradients "c", as well as the current gradients "this" on which this method is invoked. 
// Classic momentum (unitGainFactor == 1.0):
// 1) c = momentum * c + this
// Unit-gain momentum (unitGainFactor == 1.0 - momentum):
// 1) c = momentum * c + (1.0 - momentum) * this
// 2) this = c
// TODO: NormalGrad is a misnomer here. Come up with a better name.
// Updates a dense matrix.
// TODO: this should be const.
template <class ElemType>
void GPUSparseMatrix<ElemType>::NormalGrad(GPUMatrix<ElemType>& c, const ElemType momentum, ElemType unitGainFactor)
{
    VerifyWritable(__FUNCTION__);

    if (c.IsEmpty())
    {
        c.RequireSize(GetNumRows(), GetNumCols());
        c.SetValue(0.0);
    }

    if (GetFormat() == matrixFormatSparseBlockCol || GetFormat() == matrixFormatSparseBlockRow)
    {
        bool isBlockCol = (GetFormat() == MatrixFormat::matrixFormatSparseBlockCol);
        SyncGuard syncGuard;
        LONG64 N = (LONG64) GetNumNZElements();
        int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);

        _normalGradForSparseBlock<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(
            momentum,
            isBlockCol,
            GetNumRows(),
            GetNumCols(),
            GetBlockSize(),
            Data(),
            BlockId2ColOrRow(),
            c.Data(),
            unitGainFactor);
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

// -> dense
// TODO: this should be const
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::Adagrad(GPUMatrix<ElemType>& c, const bool needAveMultiplier)
{
    VerifyWritable(__FUNCTION__);

    size_t numColsNeeded = GetNumCols();
    if (needAveMultiplier)
        numColsNeeded += GetNumCols();

    if (c.IsEmpty() || c.GetNumCols() < numColsNeeded)
    {
        c.RequireSize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);
    }

    assert(c.GetNumRows() == GetNumRows() && c.GetNumCols() == numColsNeeded);

    size_t n = this->GetNumElements();

    ElemType* multipliers = nullptr;
    if (needAveMultiplier)
        multipliers = c.Buffer() + n; // temp memory used to store multipliers,

    if (GetFormat() == MatrixFormat::matrixFormatSparseCSC || GetFormat() == MatrixFormat::matrixFormatSparseCSR)
    {
        NOT_IMPLEMENTED;
    }
    else if (GetFormat() == MatrixFormat::matrixFormatSparseBlockCol || GetFormat() == MatrixFormat::matrixFormatSparseBlockRow)
    {
        let nz = NzCount();
        int blocksPerGrid = (nz + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
        bool colMajor = GetFormat() == MatrixFormat::matrixFormatSparseBlockCol;
        size_t len = colMajor ? GetNumRows() : GetNumCols();
        _adagrad4BlockSparse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(c.Buffer(), c.GetNumRows(), Data(), BlockId2ColOrRow(), multipliers, colMajor, len, nz);
    }
    else
        NOT_IMPLEMENTED;

    if (!needAveMultiplier)
        return 1;

    let nz = NzCount();
    hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float aveMultiplier = 0;
        CUBLAS_CALL(hipblasSasum(cuHandle, (LONG64) nz, reinterpret_cast<float*>(multipliers), 1, &aveMultiplier));
        return (ElemType) aveMultiplier / nz;
    }
    else
    {
        double aveMultiplier = 0;
        CUBLAS_CALL(hipblasDasum(cuHandle, (LONG64) nz, reinterpret_cast<double*>(multipliers), 1, &aveMultiplier));
        return (ElemType) aveMultiplier / nz;
    }
}

// -> dense
// TODO: This should be const
template <class ElemType>
void GPUSparseMatrix<ElemType>::FSAdagrad(
    GPUMatrix<ElemType>& c,
    GPUMatrix<ElemType>& functionValues,
    ElemType learnRatePerSample,
    ElemType momentum,
    ElemType adaWeight,
    ElemType adaMul,
    ElemType unitGainFactor)
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseBlockCol)
    {
        NOT_IMPLEMENTED;
    }

    size_t numColsNeeded = 2 * GetNumCols();

    if (c.IsEmpty() || (c.GetNumCols() < numColsNeeded))
    {
        c.RequireSize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);
    }

    assert((c.GetNumRows() == GetNumRows()) && (c.GetNumCols() == numColsNeeded));

    size_t n = GetNumElements();
    int blocksPerGrid = (n + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
    _fsadagrad4BlockSparseCol<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock >> >(
        n, Data(), ColOrRow2BlockId(), GetNumRows(),
        c.Data(), c.Data() + n, functionValues.Data(),
        learnRatePerSample, momentum, adaWeight, adaMul, unitGainFactor);
}

// -> dense
// TODO: This should be const
template <class ElemType>
void GPUSparseMatrix<ElemType>::Adam(
    GPUMatrix<ElemType>& c,
    GPUMatrix<ElemType>& functionValues,
    ElemType learnRatePerSample,
    ElemType momentum,
    ElemType adaWeight,
    ElemType adaMul,
    ElemType epsilon,
    ElemType unitGainFactor,
    bool adamax)
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseBlockCol)
    {
        NOT_IMPLEMENTED;
    }

    size_t numColsNeeded = 2 * GetNumCols();

    if (c.IsEmpty() || (c.GetNumCols() < numColsNeeded))
    {
        c.RequireSize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);
    }

    assert((c.GetNumRows() == GetNumRows()) && (c.GetNumCols() == numColsNeeded));

    size_t n = GetNumElements();
    int blocksPerGrid = (n + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
    _adam4BlockSparseCol<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock >> >(
        n, Data(), ColOrRow2BlockId(), GetNumRows(),
        c.Data(), c.Data() + n, functionValues.Data(),
        learnRatePerSample, momentum, adaWeight, adaMul, epsilon, unitGainFactor, adamax);
}

// -> dense
// TODO: This should be const
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::RmsProp(GPUMatrix<ElemType>& c,
    ElemType RMS_GAMMA,
    ElemType RMS_WGT_INC,
    ElemType RMS_WGT_MAX,
    ElemType RMS_WGT_DEC,
    ElemType RMS_WGT_MIN,
    const bool needAveMultiplier,
    const bool initialized)
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseBlockCol)
    {
        NOT_IMPLEMENTED;
    }

    const ElemType floor = 1e-6f;
    static ElemType* upd_gpu = (ElemType*)0;

    size_t n = GetNumElements();
    int blocksPerGrid = (c.GetNumElements() + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;

    size_t numColsNeeded = GetNumCols() * 3;
    if (needAveMultiplier)
        numColsNeeded += GetNumCols();

    if (c.IsEmpty() || c.GetNumCols() < numColsNeeded || !initialized)
    {
        c.RequireSize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);

        ElemType* avars = c.Data();         // accumulated variances for RMS scaling
        ElemType* signs = c.Data() + n;     // sign of previous gradient
        ElemType* steps = c.Data() + 2 * n; // current step size
                                            // Data()+3*n is temp memory used to store multipliers, no need to initialize

        _rmsprop_init4BlockSparseCol<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock >> >(
            avars, signs, steps, 
            Data(), ColOrRow2BlockId(), GetNumRows(),
            n);
    }
    assert(c.GetNumRows() == GetNumRows() && c.GetNumCols() == numColsNeeded);

    ElemType* avars = c.Data();         // accumulated variances for RMS scaling
    ElemType* signs = c.Data() + n;     // sign of previous gradient
    ElemType* steps = c.Data() + 2 * n; // current step size

    ElemType* multipliers = nullptr;
    if (needAveMultiplier)
        multipliers = c.Data() + 3 * n; // temp memory used to store multipliers,

    if (!upd_gpu)
    {
        const ElemType upd[] = {
            2, 2, 0,
            2, 2, 0,
            1, 1, 1,
            2, 2, 0,
            1, 2, 1,
            0, 2, 2,
            1, 1, 1,
            0, 2, 2,
            0, 2, 2,
        };

        upd_gpu = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 27);
        CUDA_CALL(hipMemcpy(upd_gpu, upd, sizeof(ElemType) * _countof(upd), hipMemcpyHostToDevice));
    }

    _rmsprop4BlockSparseCol<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock >> >(
        avars, signs, steps,
        Data(), ColOrRow2BlockId(), GetNumRows(),
        n,
        RMS_GAMMA, RMS_WGT_INC, RMS_WGT_MAX, RMS_WGT_DEC, RMS_WGT_MIN,
        floor, upd_gpu, multipliers);

    if (!needAveMultiplier)
        return 1;

    hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float aveMultiplier = 0;
        CUBLAS_CALL(hipblasSasum(cuHandle, (CUDA_LONG)n, reinterpret_cast<float*>(multipliers), 1, &aveMultiplier));
        return aveMultiplier / n;
    }
    else
    {
        double aveMultiplier = 0;
        CUBLAS_CALL(hipblasDasum(cuHandle, (CUDA_LONG)n, reinterpret_cast<double*>(multipliers), 1, &aveMultiplier));
        return (ElemType)aveMultiplier / n;
    }
}

// -> dense
// TODO: This should be const
template <class ElemType>
void GPUSparseMatrix<ElemType>::AdaDelta(GPUMatrix<ElemType>&c, GPUMatrix<ElemType>&functionValues, ElemType learningRate, ElemType rho, ElemType epsilon)
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseBlockCol)
    {
        NOT_IMPLEMENTED;
    }

    size_t numColsNeeded = 2 * GetNumCols();

    if (c.IsEmpty() || (c.GetNumCols() < numColsNeeded))
    {
        c.RequireSize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);
    }

    assert((c.GetNumRows() == GetNumRows()) && (c.GetNumCols() == numColsNeeded));

    size_t n = GetNumElements();
    int blocksPerGrid = (n + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
    _adadelta4BlockSparseCol<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock >> >(
        n, Data(), ColOrRow2BlockId(), GetNumRows(),
        c.Data(), c.Data() + n, functionValues.Data(),
        learningRate, rho, epsilon);
}

// sparse X dense = dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, const bool transposeA,
                                                       const GPUMatrix<ElemType>& b, const bool transposeB, ElemType beta, GPUMatrix<ElemType>& c)
{
    if (transposeB)
        NOT_IMPLEMENTED;

    // Note: This function is written for 'a' being in CSR format. If 'a' is CSC, we reinterpret it as CSR by transposing it.
    if (a.GetFormat() != matrixFormatSparseCSR && a.GetFormat() != matrixFormatSparseCSC)
        NOT_IMPLEMENTED;
    const bool reinterpretAsCSR = a.GetFormat() == matrixFormatSparseCSC;

    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || (b.GetComputeDeviceId() != a.GetComputeDeviceId()))
        RuntimeError("MultiplyAndWeightedAdd: All matrices must be on the same GPU");

    a.PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseOperation_t oper = (transposeA != reinterpretAsCSR) ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

    int n = (int)b.GetNumCols();
    int m = (int)(reinterpretAsCSR ? a.GetNumCols() : a.GetNumRows());
    int k = (int)(reinterpretAsCSR ? a.GetNumRows() : a.GetNumCols());
    assert(n == (int) c.GetNumCols());

    const auto& aRowLocation = reinterpretAsCSR ? a.ColLocation() : a.RowLocation();
    const auto& aColLocation = reinterpretAsCSR ? a.RowLocation() : a.ColLocation();

    SyncGuard syncGuard;
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseScsrmm(cusparseHandle, oper, m, n, k, (int) a.GetNumNZElements(), reinterpret_cast<float*>(&alpha), descr, reinterpret_cast<const float*>(a.Buffer()),
                                     aRowLocation, aColLocation, reinterpret_cast<float*>(b.Data()),
                                     (int) b.GetNumRows(), reinterpret_cast<float*>(&beta), reinterpret_cast<float*>(c.Data()), (int) c.GetNumRows()));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsrmm(cusparseHandle, oper, m, n, k, (int) a.GetNumNZElements(), reinterpret_cast<double*>(&alpha), descr, reinterpret_cast<const double*>(a.Buffer()),
                                     aRowLocation, aColLocation, reinterpret_cast<double*>(b.Data()),
                                     (int) b.GetNumRows(), reinterpret_cast<double*>(&beta), reinterpret_cast<double*>(c.Data()), (int) c.GetNumRows()));
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S, const GPUMatrix<ElemType>& D, GPUMatrix<ElemType>& C)
{
    C.RequireSize(S.GetNumRows(), D.GetNumCols());

    MultiplyAndWeightedAdd(1, S, false, D, false, 0, C);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& D, const GPUSparseMatrix<ElemType>& S, GPUMatrix<ElemType>& C)
{
    C.RequireSize(S.GetNumCols(), D.GetNumRows());

    MultiplyAndWeightedAdd(1, D, false, S, false, 0, C);
}

// ElemCountFromBufferSize - Return the elemCountAllocated for a particular buffersize
// totalBufferSize - total buffer we have to use
// return: size of allocated elements/index slots available
template <class ElemType>
size_t GPUSparseMatrix<ElemType>::ElemCountFromBufferSize(const size_t numRows, const size_t numCols, const MatrixFormat format, const size_t totalBufferSize) const
{
    size_t elemSizeAllocated;
    if (format == matrixFormatSparseCSC)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * (numCols + 1)) / (sizeof(GPUSPARSE_INDEX_TYPE) + sizeof(ElemType));
    }
    else if (format == matrixFormatSparseCSR)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * (numRows + 1)) / (sizeof(GPUSPARSE_INDEX_TYPE) + sizeof(ElemType));
    }
    else if (format == matrixFormatSparseBlockCol)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * 2 * numCols) / sizeof(ElemType);
    }
    else if (format == matrixFormatSparseBlockCol || format == matrixFormatSparseBlockRow)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * 2 * numRows) / sizeof(ElemType);
    }
    else // uncompressed COO format
    {
        elemSizeAllocated = totalBufferSize / (2 * sizeof(GPUSPARSE_INDEX_TYPE) + sizeof(ElemType));
    }
    return elemSizeAllocated;
}

template <class ElemType>
size_t GPUSparseMatrix<ElemType>::ElemCountFromBufferSize() const
{
    return ElemCountFromBufferSize(GetNumRows(), GetNumCols(), GetFormat(), BufferSizeAllocated());
}

// PrepareBuffer - Get the dimensions start buffer, computes the starting row/column of each value
// m - rows in the source
// n - cols in the source
// canReuseBuffer - target matrix can be reused for temporary space
// func - function to call to count elements in the result (returns count, and fills csrRowPtr array)
template <class ElemType>
void GPUSparseMatrix<ElemType>::PrepareBuffer(size_t m, size_t n, bool canReuseBuffer, std::function<size_t(GPUSPARSE_INDEX_TYPE* csrRowPtrC)> func)
{
    VerifyWritable(__FUNCTION__);

    if (this->GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    PrepareDevice();

    GPUSPARSE_INDEX_TYPE* csrRowPtrC = nullptr;
    GPUSparseMatrix<ElemType>& c = *this;
    size_t cSize = c.BufferSizeAllocated();
    size_t rowBufferRequired = (m + 1) * sizeof(GPUSPARSE_INDEX_TYPE);
    bool allocatedBuffer = false;

    // do we have enough memory to store just the row buffer?
    // BUGBUG? I suspect Data() here should be Buffer().
    if (cSize >= rowBufferRequired && c.Data_IThinkThisShouldBeBuffer() != nullptr && canReuseBuffer)
    {
        // BUGBUG? I suspect Data() here should be Buffer().
        csrRowPtrC = (GPUSPARSE_INDEX_TYPE*) c.Data_IThinkThisShouldBeBuffer();
    }
    else
    {
        csrRowPtrC = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(GetComputeDeviceId(), rowBufferRequired / sizeof(GPUSPARSE_INDEX_TYPE));
        allocatedBuffer = true;
    }

    // get the non-zero count from the function (and
    size_t nnzC = func(csrRowPtrC);

    // now we know the number of Non-zeros in the result set, set the output size
    c.RequireSizeAndAllocate(m, n, nnzC, true, false);

    CUDA_CALL(hipMemcpy(c.SecondaryIndexLocation(), csrRowPtrC, c.SecondaryIndexSize(), hipMemcpyDeviceToDevice));
    VerifyCachedNzCount(nnzC); // (to be sure)

    // if we allocated the buffer, free it here
    if (allocatedBuffer)
        TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(GetComputeDeviceId(), csrRowPtrC);
}

// Multiply - multiply one sparse matrix by another sparse matrix
// S1 - first sparse matrix
// transposeS1 - transpose first matrix?
// S2 - second sparse matrix
// transposeS2 - tanspose second matrix?
// c - result matrix
// NOTE: if c has enough space allocated, it will be reused, otherwise it will be freed and a new memory block used
template <class ElemType>
void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S1, bool transposeS1, const GPUSparseMatrix<ElemType>& S2, bool transposeS2, 
                                         GPUSparseMatrix<ElemType>& c)
{
    c.VerifyWritable(__FUNCTION__);

    if (S1.GetFormat() != matrixFormatSparseCSR || S2.GetFormat() != matrixFormatSparseCSR || c.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (S1.GetComputeDeviceId() != S2.GetComputeDeviceId())
        RuntimeError("Sparse matrix multiply: both matrices must be on the same device");

    S1.PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrB));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrC));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseOperation_t operA = transposeS1 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t operB = transposeS2 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

    int m = int(transposeS1 ? S1.GetNumCols() : S1.GetNumRows());
    int n = int(transposeS2 ? S2.GetNumRows() : S2.GetNumCols());
    int k = int(transposeS1 ? S1.GetNumRows() : S1.GetNumCols());
    int l = int(transposeS2 ? S2.GetNumCols() : S2.GetNumRows());
    if (k != l)
        RuntimeError("Sparse matrix multiply: dimensionality mismatch");

    int nnzA = (int) S1.GetNumNZElements();
    int nnzB = (int) S2.GetNumNZElements();

    SyncGuard syncGuard;
    // Step 1
    c.PrepareBuffer(m, n, false, // false means we cannot reuse the "c" buffer if it exists for temporaries
                    [&](GPUSPARSE_INDEX_TYPE* csrRowPtrC) -> size_t
                    {
                        int nnzTotal = -1;
                        CUSPARSE_CALL(hipsparseXcsrgemmNnz(cusparseHandle, operA, operB, m, n, k, descrA, nnzA, S1.RowLocation(), S1.ColLocation(), descrB, nnzB,
                                                          S2.RowLocation(), S2.ColLocation(), descrC, csrRowPtrC, &nnzTotal));
                        return nnzTotal;
                    });

    // Step 2
    if (sizeof(float) == sizeof(ElemType))
    {
        CUSPARSE_CALL(hipsparseScsrgemm(cusparseHandle, operA, operB, m, n, k, descrA, nnzA, (const float*) S1.Buffer(), S1.RowLocation(), S1.ColLocation(),
                                       descrB, nnzB, (const float*) S2.Buffer(), S2.RowLocation(), S2.ColLocation(),
                                       // BUGBUG? I suspect Data() here should be Buffer().
                                       descrC, (float*) c.Data_IThinkThisShouldBeBuffer(), c.RowLocation(), c.ColLocation()));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsrgemm(cusparseHandle, operA, operB, m, n, k, descrA, nnzA, (const double*) S1.Buffer(), S1.RowLocation(), S1.ColLocation(),
                                       descrB, nnzB, (const double*) S2.Buffer(), S2.RowLocation(), S2.ColLocation(),
                                       // BUGBUG? I suspect Data() here should be Buffer().
                                       descrC, (double*) c.Data_IThinkThisShouldBeBuffer(), c.RowLocation(), c.ColLocation()));
    }
    hipsparseDestroy(cusparseHandle);
    c.VerifyCachedNzCount(c.NzCount()); // (to be sure)
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignProductOf(const GPUSparseMatrix<ElemType>& a, const bool transposeA, const GPUSparseMatrix<ElemType>& b, const bool transposeB)
{
    Multiply(a, transposeA, b, transposeB, *this);
    return *this;
}

// sparse op sparse -> sparse
template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUSparseMatrix<ElemType>& c)
{
    if (a.GetFormat() != matrixFormatSparseCSR || b.GetFormat() != matrixFormatSparseCSR )
    {
        NOT_IMPLEMENTED;
    }
    if (c.m_sob.get() == nullptr)
        c.ZeroInit(a.GetFormat(), a.GetComputeDeviceId());

    if (a.GetNumCols() != b.GetNumCols() || a.GetNumRows() != b.GetNumRows())
        RuntimeError("Dimensions mismatch in ScaleAndAdd");
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId())
        RuntimeError("ScaleAndAdd: matrices must be on the same device");

    c.SetFormat(a.GetFormat());
    c.SetComputeDeviceId(a.GetComputeDeviceId());
    int m = (int) a.GetNumRows();
    int n = (int) a.GetNumCols();
    int nnzA = (int) a.GetNumNZElements();
    int nnzB = (int) b.GetNumNZElements();

    a.PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrB));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrC));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);

    SyncGuard syncGuard;
    // Step 1
    bool inOutParameter = (&b == &c);
    c.PrepareBuffer(m, n, !inOutParameter, 
                    [&](GPUSPARSE_INDEX_TYPE* csrRowPtrC) -> size_t
                    {
                        int nnzTotal = -1;
                        CUSPARSE_CALL(hipsparseXcsrgeamNnz(cusparseHandle, m, n, descrA, nnzA, a.RowLocation(), a.ColLocation(), descrB, nnzB, b.RowLocation(), b.ColLocation(), descrC, csrRowPtrC, &nnzTotal));
                        return nnzTotal;
                    });

    // Step 2
    if (sizeof(ElemType) == sizeof(float))
    {
        // BUGBUG? I suspect Data() here should be Buffer().
        CUSPARSE_CALL(hipsparseScsrgeam(cusparseHandle, m, n, reinterpret_cast<const float*>(&alpha), descrA, nnzA, reinterpret_cast<const float*>(a.Data_IThinkThisShouldBeBuffer()), a.RowLocation(), a.ColLocation(),
                                       reinterpret_cast<const float*>(&beta), descrB, nnzB, reinterpret_cast<const float*>(b.Data_IThinkThisShouldBeBuffer()), b.RowLocation(), b.ColLocation(), descrC, reinterpret_cast<float*>(c.Data_IThinkThisShouldBeBuffer()), c.RowLocation(), c.ColLocation()));
    }
    else
    {
        // BUGBUG? I suspect Data() here should be Buffer().
        CUSPARSE_CALL(hipsparseDcsrgeam(cusparseHandle, m, n, reinterpret_cast<const double*>(&alpha), descrA, nnzA, reinterpret_cast<const double*>(a.Data_IThinkThisShouldBeBuffer()), a.RowLocation(), a.ColLocation(),
                                       reinterpret_cast<const double*>(&beta), descrB, nnzB, reinterpret_cast<const double*>(b.Data_IThinkThisShouldBeBuffer()), b.RowLocation(), b.ColLocation(), descrC, reinterpret_cast<double*>(c.Data_IThinkThisShouldBeBuffer()), c.RowLocation(), c.ColLocation()));
    }
    hipsparseDestroy(cusparseHandle);
    c.VerifyCachedNzCount(c.NzCount()); // (to be sure)
}

// sparse op dense -> dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    if (a.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (a.GetNumRows() != b.GetNumRows() || a.GetNumRows() != c.GetNumRows() || a.GetNumCols() != b.GetNumCols() || a.GetNumCols() != c.GetNumCols())
        LogicError("ScaleAndAdd: dimension mismatch");
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || a.GetComputeDeviceId() != c.GetComputeDeviceId())
        RuntimeError("ScaleAndAdd: matrices must be on the same device");
    b.PrepareDevice();
    // copy b to c
    CUDA_CALL(hipMemcpy(c.Data(), b.Data(), sizeof(ElemType) * b.GetNumElements(), hipMemcpyDeviceToDevice));
    if (beta != 1)
    {
        c *= beta;
    }
    SyncGuard syncGuard;
    CUDA_LONG M = (CUDA_LONG) a.GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    // BUGBUG? I suspect a.Data() here should be Buffer().
    _sparseCSRPlusDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(alpha, a.Data_IThinkThisShouldBeBuffer(), a.RowLocation(), a.ColLocation(), c.Data(), M);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    ScaleAndAdd(beta, b, alpha, a, c);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Scale(ElemType alpha, GPUSparseMatrix<ElemType>& a)
{
    a.VerifyWritable(__FUNCTION__);

    if (a.IsEmpty())
        return;

    CUDA_LONG N = (CUDA_LONG) a.GetNumNZElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _scaleArray<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(alpha, a.NzValues(), N);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ElementWisePower(ElemType alpha, const GPUSparseMatrix<ElemType>& a, GPUSparseMatrix<ElemType>& c)
{
    c.VerifyWritable(__FUNCTION__);

    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        if (a.IsEmpty())
            LogicError("ElementWisePower:  The input matrix a is empty.");

        c.ResizeAsAndCopyIndexFrom(a);

        SyncGuard syncGuard;
        a.PrepareDevice();
        CUDA_LONG N = (CUDA_LONG) a.GetNumNZElements();
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        _elementWisePowerOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(alpha, a.NzValues(), c.NzValues(), N);
    }
}

// sparse x dense = scalar
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.GetFormat() != matrixFormatSparseCSR && a.GetFormat() != matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    if (a.GetComputeDeviceId() != b.GetComputeDeviceId())
        RuntimeError("a and b must be on the same device");

    int m = (int) a.GetNumRows();
    int n = (int) a.GetNumCols();
    int nnz = (int) a.GetNumNZElements();

    ElemType* cscValA = nullptr;
    GPUSPARSE_INDEX_TYPE* cscRowIndA = nullptr;
    GPUSPARSE_INDEX_TYPE* cscColPtrA = nullptr;

    hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
    hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));

    bool allocTemp = (a.GetFormat() == matrixFormatSparseCSR);

    if (allocTemp) // need to put a in ColumnMajor format
    {
        cscValA = TracingGPUMemoryAllocator::Allocate<ElemType>(a.GetComputeDeviceId(), nnz);
        cscRowIndA = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), nnz);
        cscColPtrA = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), (n + 1));

        SyncGuard syncGuard;
        if (sizeof(ElemType) == sizeof(float))
        {
            // BUGBUG? I suspect Data() here should be Buffer().
            CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const float*>(a.Data_IThinkThisShouldBeBuffer()), a.RowLocation(), a.ColLocation(), reinterpret_cast<float*>(cscValA), cscRowIndA, cscColPtrA, cpVals, idxBase));
        }
        else
        {
            // BUGBUG? I suspect Data() here should be Buffer().
            CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const double*>(a.Data_IThinkThisShouldBeBuffer()), a.RowLocation(), a.ColLocation(), reinterpret_cast<double*>(cscValA), cscRowIndA, cscColPtrA, cpVals, idxBase));
        }
    }
    else if (a.GetFormat() == matrixFormatSparseCSC)
    {
        // BUGBUG? I suspect Data() here should be Buffer().
        cscValA = (ElemType*) a.Data_IThinkThisShouldBeBuffer();
        cscRowIndA = a.RowLocation();
        cscColPtrA = a.ColLocation();
    }
    else
    {
        NOT_IMPLEMENTED;
    }
    let a_nz = a.NzCount();
    // Given sparse matrix in column major format, calculate indices for corresponding sparse vector
    GPUSPARSE_INDEX_TYPE* vectArray = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), a_nz);
    CUDA_LONG M = n;
    CUDA_LONG N = m;
    // GPUSPARSE_INDEX_TYPE* h_vectArray= new int[a.m_nz];
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _getSparseVectorRepresntationForCSCMatrix<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(cscColPtrA, cscRowIndA, vectArray, M, N);
    if (allocTemp)
    {
        TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), cscRowIndA);
        TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), cscColPtrA);
    }
    // CUDA_CALL(hipMemcpy(h_vectArray,vectArray,sizeof(GPUSPARSE_INDEX_TYPE)*a.m_nz,hipMemcpyDeviceToHost));

    // Actual dot product
    ElemType res = 0;
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseSdoti(cusparseHandle, (int) a_nz, reinterpret_cast<float*>(cscValA), vectArray,
                                    reinterpret_cast<float*>(b.Data()),
                                    reinterpret_cast<float*>(&res), idxBase));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDdoti(cusparseHandle, (int) a_nz, reinterpret_cast<double*>(cscValA), vectArray,
                                    reinterpret_cast<double*>(b.Data()),
                                    reinterpret_cast<double*>(&res), idxBase));
    }
    TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), vectArray);
    if (allocTemp)
    {
        TracingGPUMemoryAllocator::Free<ElemType>(a.GetComputeDeviceId(), cscValA);
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
    return res;
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
{
    return GPUSparseMatrix<ElemType>::InnerProductOfMatrices(b, a);
}

// sparse op dense -> dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::InnerProduct(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c, const bool isColWise)
{
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId()) // different GPUs
        InvalidArgument("All matrices must be on the same GPU");

    if (a.IsEmpty() || b.IsEmpty())
        LogicError("Scale:  one of the input matrices is empty.");

    if (a.GetFormat() != MatrixFormat::matrixFormatSparseCSC)
    {
        NOT_IMPLEMENTED;
    }

    const int m = (int)a.GetNumRows();
    const int n = (int)a.GetNumCols();
    const int k = (int)b.GetNumRows();
    const int l = (int)b.GetNumCols();

    assert(m > 0 && n > 0 && k > 0 && l > 0); // converting from size_t to int may cause overflow
    assert(m == k && n == l);                 // converting from size_t to int may cause overflow
    if (m != k || n != l)
        InvalidArgument("Matrices a and b should have same dimension.");

    if (isColWise)
        c.RequireSize(1, n);
    else
        c.RequireSize(m, 1);

    c.PrepareDevice();

    int blocksPerGrid = 0;
    if (isColWise) // col-wise
    {
        blocksPerGrid = (int)ceil(1.0 * n / GridDim::maxThreadsPerBlock);
    }
    else
    {
        blocksPerGrid = (int)ceil(1.0 * m / GridDim::maxThreadsPerBlock);
    }

    SyncGuard syncGuard;
    _innerProduct4SparseCSC<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
        c.Data(),
        a.Buffer(), a.RowLocation(), a.ColLocation(),
        b.Data(),
        m, n, isColWise);
}

// This is an utility function useful for debugging issues with sparse matrices.
// It just checks that the CSC format indices are not corrupted / pointing to invalid memory.
template <class ElemType>
bool GPUSparseMatrix<ElemType>::IsValid() const
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    long* res = new long[4];
    res[0] = 1;
    res[1] = 0;
    res[2] = 0;
    res[3] = 0;
    long* d_res = TracingGPUMemoryAllocator::Allocate<long>(GetComputeDeviceId(), 4);
    CUDA_CALL(hipMemcpy(d_res, res, sizeof(long) * 4, hipMemcpyHostToDevice));

    SyncGuard syncGuard;
    int blocksPerGrid = (int) ceil((1.0 * SecondaryIndexCount()) / GridDim::maxThreadsPerBlock);
    _isValid<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(MajorIndexLocation(), SecondaryIndexLocation(), GetNumRows(), GetNumCols(), GetNumNZElements(), d_res);

    CUDA_CALL(hipMemcpy(res, d_res, sizeof(long) * 4, hipMemcpyDeviceToHost));

    if (res[0] == 1)
    {
        return true;
    }
    else
    {
        fprintf(stderr, "GPUSparseMatrix::IsValid returned false (additional info: %ld %ld %ld %ld)\n", res[0], res[1], res[2], res[3]);
        return false;
    }
}

template <class ElemType>
/*static*/ bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b,
                                                    const ElemType threshold)
{
    if (a.GetNumNZElements() != b.GetNumNZElements() || a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;

    if (a.GetFormat() != b.GetFormat())
        NOT_IMPLEMENTED;

    long* res = new long[3];
    res[0] = 1;
    res[1] = 1;
    res[2] = 1;
    long* d_res = TracingGPUMemoryAllocator::Allocate<long>(a.GetComputeDeviceId(), 3);
    CUDA_CALL(hipMemcpy(d_res, res, sizeof(long) * 3, hipMemcpyHostToDevice));

    int blocksPerGrid = (int) ceil(1.0 * a.GetNumNZElements() / GridDim::maxThreadsPerBlock);
    _areEqual<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.NzValues(), b.NzValues(), (CUDA_LONG) a.GetNumNZElements(), threshold, d_res);
    _areEqual<int><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.MajorIndexLocation(), b.MajorIndexLocation(), (CUDA_LONG) a.MajorIndexCount(), (int) threshold, d_res + 1);
    blocksPerGrid = (int) ceil((1.0 * a.SecondaryIndexCount()) / GridDim::maxThreadsPerBlock);
    _areEqual<int><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.SecondaryIndexLocation(), b.SecondaryIndexLocation(), (CUDA_LONG) a.SecondaryIndexCount(), (int) threshold, d_res + 2);

    CUDA_CALL(hipMemcpy(res, d_res, sizeof(long) * 3, hipMemcpyDeviceToHost));
    if (res[0] * res[1] * res[2] == 1)
        return true;
    else
        return false;
}

template <class ElemType>
/*static*/ bool GPUSparseMatrix<ElemType>::AreEqual(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b,
                                                    const ElemType threshold)
{
    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;
    GPUSparseMatrix<ElemType> c(b.GetComputeDeviceId(), b.GetFormat());
    c.SetValue(a);
    return AreEqual(c, b, threshold);
}

template <class ElemType>
/*static*/ bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b,
                                                    const ElemType threshold)
{
    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;
    GPUSparseMatrix<ElemType> c(a.GetComputeDeviceId(), a.GetFormat());
    c.SetValue(b);
    return AreEqual(a, c, threshold);
}

template <class ElemType>
bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUSparseMatrix<ElemType>& a, const ElemType threshold) const
{
    return AreEqual(*this, a, threshold);
}

template <class ElemType>
bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUMatrix<ElemType>& a, const ElemType threshold) const
{
    return AreEqual(*this, a, threshold);
}

#pragma endregion Static BLAS Functions

#pragma region Member BLAS Functions

// sparse x dense = dense
template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        LogicError("ElementProductOf: matrix dimensions mismatch");

    b.PrepareDevice();
    GPUMatrix<ElemType> c(b.GetNumRows(), b.GetNumCols(), b.GetComputeDeviceId());

    SyncGuard syncGuard;
    CUDA_LONG M = (CUDA_LONG) a.GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    // BUGBUG? I suspect a.Data() here should be Buffer().
    _sparseCSRElemMulDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.Data_IThinkThisShouldBeBuffer(), a.RowLocation(), a.ColLocation(), b.Data(), c.Data(), M);
    return c;
}

// sparse x dense = dense
template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
{
    return GPUSparseMatrix<ElemType>::ElementProductOf(b, a);
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator+(const GPUSparseMatrix<ElemType>& a) const
{
    GPUSparseMatrix<ElemType> res(GetComputeDeviceId(), GetFormat());
    GPUSparseMatrix<ElemType>::ScaleAndAdd(1, *this, 1, a, res);
    return res;
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator-(const GPUSparseMatrix<ElemType>& a) const
{
    GPUSparseMatrix<ElemType> res(GetComputeDeviceId(), GetFormat());
    GPUSparseMatrix<ElemType>::ScaleAndAdd(1, *this, -1, a, res);
    return res;
}

// TODO: This is an unusual use of this operator. Remove this.
template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator^=(ElemType alpha)
{
    GPUSparseMatrix<ElemType>& us = *this;
    ElementWisePower(alpha, us, us);
    return us;
}

// TODO: This is an unusual use of this operator. Remove this.
template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator^(ElemType alpha) const
{
    GPUSparseMatrix<ElemType> c(GetComputeDeviceId(), GetFormat());
    ElementWisePower(alpha, *this, c);
    return c;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator*=(ElemType alpha)
{
    GPUSparseMatrix<ElemType>& us = *this;
    if (alpha != 1)
        Scale(alpha, us);
    return us;
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator*(ElemType alpha) const
{
    GPUSparseMatrix<ElemType> c(*this);
    if (alpha != 1)
        Scale(alpha, c);
    return c;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementPowerOf(const GPUSparseMatrix<ElemType>& a, const ElemType power)
{
    ElementWisePower(power, a, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::Transpose() const
{
    int m = (int) GetNumRows();
    int n = (int) GetNumCols();
    int nnz = (int) GetNumNZElements();
    hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
    hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

    assert(GetFormat() & matrixFormatCompressed); // for now this only supports compressed formats
    PrepareDevice();
    GPUSparseMatrix c(GetComputeDeviceId(), GetFormat());
    c.RequireSizeAndAllocate(n, m, nnz, GetFormat(), true, false);

    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));

    SyncGuard syncGuard;
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSR)
    {
        if (nnz > 0)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                // BUGBUG? I suspect Data() here should be Buffer().
                CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const float*>(Data_IThinkThisShouldBeBuffer()), RowLocation(), ColLocation(),
                                               reinterpret_cast<float*>(c.Data_IThinkThisShouldBeBuffer()), c.ColLocation(), c.RowLocation(), cpVals, idxBase));
            }
            else
            {
                // BUGBUG? I suspect Data() here should be Buffer().
                CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const double*>(Data_IThinkThisShouldBeBuffer()), RowLocation(), ColLocation(),
                                               reinterpret_cast<double*>(c.Data_IThinkThisShouldBeBuffer()), c.ColLocation(), c.RowLocation(), cpVals, idxBase));
            }
        }
        else
        {
            CUDA_CALL(hipMemset(c.Buffer(), 0, c.BufferSizeAllocated()));
        }
    }
    else if (GetFormat() == matrixFormatSparseCSC)
    {
        if (nnz > 0)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                // BUGBUG? I suspect Data() here should be Buffer().
                CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, n, m, nnz, reinterpret_cast<const float*>(this->Data_IThinkThisShouldBeBuffer()), this->ColLocation(), this->RowLocation(),
                                               reinterpret_cast<float*>(c.Data_IThinkThisShouldBeBuffer()), c.RowLocation(), c.ColLocation(), cpVals, idxBase));
            }
            else
            {
                // BUGBUG? I suspect Data() here should be Buffer().
                CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, n, m, nnz, reinterpret_cast<const double*>(this->Data_IThinkThisShouldBeBuffer()), this->ColLocation(), this->RowLocation(),
                                               reinterpret_cast<double*>(c.Data_IThinkThisShouldBeBuffer()), c.RowLocation(), c.ColLocation(), cpVals, idxBase));
            }
        }
        else
        {
            CUDA_CALL(hipMemset(c.Buffer(), 0, c.BufferSizeAllocated()));
        }
    }
    else
    {
        NOT_IMPLEMENTED;
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
    UpdateCachedNzCount(nnz);
    return c;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTransposeOf(const GPUSparseMatrix<ElemType>& a)
{
    VerifyWritable(__FUNCTION__);

    if (this == &a)
        LogicError("AssignTransposeOf: a is the same as [this]. Does not support inplace transpose.");

    if (a.IsEmpty())
        LogicError("AssignTransposeOf: Matrix a is empty.");

    *this = a.Transpose();
    return *this;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::InplaceTranspose()
{
    if (IsEmpty())
        return;
    // transfer converted block over to this pointer
    *this = std::move(Transpose());
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::ColumnSlice(size_t startColumn, size_t numCols) const
{
    if (startColumn + numCols > GetNumCols())
        InvalidArgument("The slice (%d+%d) is out of range of the source matrix (%d).", (int) startColumn, (int) numCols, (int) GetNumCols());

    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC && (startColumn != 0 || numCols != GetNumCols()))
        NOT_IMPLEMENTED;

    GPUSparseMatrix<ElemType> slice(GetComputeDeviceId());
    slice.ShallowCopyFrom(*this);
    slice.SetNumCols(numCols);
    slice.m_sliceViewOffset = m_sliceViewOffset + startColumn; // Just shift the compressed index location to the new startColumn - that's it!
    if (startColumn == 0 && numCols == GetNumCols() && HasCachedNzCount())
        slice.UpdateCachedNzCount(NzCount());

    return slice;
}

// -> dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::AssignColumnSliceToDense(GPUMatrix<ElemType>& slice, size_t startColumn, size_t numCols) const
{
    int m = (int) GetNumRows();
    int n = (int) GetNumCols();

    // We can either error out or RequireSize. Because RequireSize will error out if it's not allowed, I think this makes more sense.
    slice.RequireSize(m, numCols);

    if (startColumn + numCols > n)
        InvalidArgument("The slice (%d+%d) is out of range of the source matrix (%d).", (int) startColumn, (int) numCols, (int) n);

    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC)
    {
        if ((startColumn != 0) || (numCols != GetNumCols()))
            NOT_IMPLEMENTED;

        return CopyToDenseMatrix(slice);
    }

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    SyncGuard syncGuard;
    CUSPARSE_CALL(hipsparseSetStream(cusparseHandle, t_stream));
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseScsc2dense(cusparseHandle, m, numCols, descr, (float*) Buffer(), RowLocation(), ColLocation() + startColumn, (float*) slice.Data(), m));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsc2dense(cusparseHandle, m, numCols, descr, (double*) Buffer(), RowLocation(), ColLocation() + startColumn, (double*) slice.Data(), m));
    }

    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));

}
template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::CopyColumnSliceToDense(size_t startColumn, size_t numCols) const
{
    GPUMatrix<ElemType> slice(GetNumRows(), numCols, GetComputeDeviceId());

    AssignColumnSliceToDense(slice, startColumn, numCols);

    return slice;
}

template<class ElemTypePtr> // ElemType* or const ElemType*
struct CSCSlice
{
    ElemTypePtr m_buffer;          // pointer to first array
    CUDA_LONG m_elemSizeAllocated; // pointer delta between the 3 arrays
    CUDA_LONG m_firstColumn;       // start of slice
    CUDA_LONG m_numColumns;        // width of slice
    // note: bad alignment, got 4 extra padding bytes free :(
};

template <size_t N, class ElemType>
__global__ void _gatherMemcpyCSC(const CSCSlice<ElemType*> outputSlice, const FixedSizeParameterArray<N, CSCSlice<const ElemType*>> inputSlices)
{
    // output data arrays
    auto* outputBuffer = outputSlice.m_buffer;
    let outputElemSizeAllocated = outputSlice.m_elemSizeAllocated;
    auto* outputRowIndices    = (CUDA_LONG*)(outputBuffer + outputElemSizeAllocated);
    auto* outputColumnOffsets = outputRowIndices          + outputElemSizeAllocated;
    // adjust for starting point
    auto* outputData = outputBuffer;
    auto jo = outputSlice.m_firstColumn;
    outputColumnOffsets += jo;
    if (jo == 0) // upon first call, the very first entry has not been initialized yet
        *outputColumnOffsets = 0;
    else // otherwise position m_firstColumn must already have been written by previous launch
    {
        let firstColumnOffset = *outputColumnOffsets; // was written during last launch
        outputData       += firstColumnOffset;
        outputRowIndices += firstColumnOffset;
    }
    // ready to write.
    // loop over input slices
    for (CUDA_LONG i = 0; i < inputSlices.size(); i++)
    {
        // get input pointers for this slice
        let& inputSlice = inputSlices[i];
        let* inputData = inputSlice.m_buffer;
        let inputElemSizeAllocated = inputSlice.m_elemSizeAllocated;
        let* inputRowIndices    = (CUDA_LONG*)(inputData + inputElemSizeAllocated);
        let* inputColumnOffsets = inputRowIndices        + inputElemSizeAllocated;
        let j0 =      inputSlice.m_firstColumn;
        let j1 = j0 + inputSlice.m_numColumns;
        auto columnOffset = inputColumnOffsets[j0];
        inputData       += columnOffset;
        inputRowIndices += columnOffset;
        // write column offsets
        auto* endOutputData = outputData;
        for (CUDA_LONG j = j0 + 1; j <= j1; j++)
        {
            let endColumnOffset = inputColumnOffsets[j];
            endOutputData += endColumnOffset - columnOffset;
            *++outputColumnOffsets = endOutputData - outputBuffer;
            columnOffset = endColumnOffset;
        }
        // copy values and row indices
        while (outputData < endOutputData)
        {
            *outputData++       = *inputData++;
            *outputRowIndices++ = *inputRowIndices++;
        }
    }
}

template <size_t N, class ElemType>
static void GatherMemcpyCSC(const CSCSlice<ElemType*>& outputSlice, const MaxFixedSizeParameterArray<CSCSlice<const ElemType*>>& inputSliceBuffer)
{
    let& inputSliceArray = (const FixedSizeParameterArray<N, CSCSlice<const ElemType*>>&)inputSliceBuffer;
    SyncGuard syncGuard;
    _gatherMemcpyCSC<N, ElemType> <<<1, 1, 0, t_stream>>>(outputSlice, inputSliceArray);
}

// GatherBatch() batches many independent inputs into one output tensor
// Only supports CSC format. Matrix must already have the output shape and correct type.
// This current implementation is not efficient for data other than one-hot.
template <class ElemType>
void GPUSparseMatrix<ElemType>::GatherBatch(size_t numInputs, const std::function<const GPUSparseMatrix<ElemType>&(size_t)>& inputs)
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        InvalidArgument("GatherBatch (sparse): Requires CSC format.");
    // TODO: NzCount() is two GPU syncs! We should cache this value CPU-side.
    if (NzCount() != 0)
        InvalidArgument("GatherBatch (sparse): The target matrix cannot have pre-existing non-zero values when being gathered into.");
    // determine necessary allocation
    PrepareDevice();
    let numRows = GetNumRows();
    size_t numCols = 0;
    size_t nz = 0; // TODO: use an upper bound, to avoid GPU sync
    for (size_t i = 0; i < numInputs; i++)
    {
        let& input = inputs(i);
        if (input.GetFormat() != MatrixFormat::matrixFormatSparseCSC)
            InvalidArgument("GatherBatch (sparse): Requires CSC format.");
        if (input.GetNumRows() != numRows)
            InvalidArgument("GatherBatch (sparse): All inputs must have the same number of rows as the output (%d).", (int)numRows);
        let inputCols = input.GetNumCols();
        nz += 1;//input.NzCount(); // TODO: double-check that this does not actually read data from the GPU, that caching works
        numCols += inputCols;
    }
    if (numCols != GetNumCols())
        InvalidArgument("GatherBatch: Total number of input columns (%d) must be equal to number of output columns (%d).",
                        (int)numCols, (int)GetNumCols());
    // allocate
    RequireSizeAndAllocate(numRows, numCols, nz, /*growOnly=*/true, /*keepExistingValues=*/false);
    // process all inputs
    MaxFixedSizeParameterArray<CSCSlice<const ElemType*>> inputSliceBuffer;
    static constexpr size_t capacity = MaxFixedSizeParameterArray<CSCSlice<const ElemType*>>::CAPACITY;
    m_sliceViewOffset = 0;
    CSCSlice<ElemType*> outputSlice =
    {
        Buffer(), (CUDA_LONG)GetSizeAllocated(), (CUDA_LONG)m_sliceViewOffset, /*numCols=*/0
    };
    for (size_t i = 0; i < numInputs; i++)
    {
        let& input = inputs(i);
        let inCols = input.GetNumCols();
        if (inCols == 0)
            continue;
        inputSliceBuffer.push_back(CSCSlice<const ElemType*>
        {
            input.Buffer(), (CUDA_LONG)input.GetSizeAllocated(), (CUDA_LONG)input.m_sliceViewOffset, (CUDA_LONG)inCols
        });
        outputSlice.m_numColumns += inCols;
        if (inputSliceBuffer.size() == inputSliceBuffer.capacity())
        {
            // flush
            GatherMemcpyCSC<capacity, ElemType>(outputSlice, inputSliceBuffer);
            inputSliceBuffer.clear();
            // advance the output range column pointer
            outputSlice.m_firstColumn += outputSlice.m_numColumns;
            outputSlice.m_numColumns = 0;
        }
    }
    let colsLeft = inputSliceBuffer.size();
    if      (colsLeft == 0) {}
    else if (colsLeft <= 1)        GatherMemcpyCSC<       1, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 2)        GatherMemcpyCSC<       2, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 4)        GatherMemcpyCSC<       4, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 8)        GatherMemcpyCSC<       8, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 16)       GatherMemcpyCSC<      16, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 24)       GatherMemcpyCSC<      24, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 32)       GatherMemcpyCSC<      32, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 48)       GatherMemcpyCSC<      48, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 64)       GatherMemcpyCSC<      64, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 96)       GatherMemcpyCSC<      96, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= 128)      GatherMemcpyCSC<     128, ElemType>(outputSlice, inputSliceBuffer);
    else if (colsLeft <= capacity) GatherMemcpyCSC<capacity, ElemType>(outputSlice, inputSliceBuffer);
    else LogicError("GatherBatch: We should have flushed inside the loop, but somehow didn't??");
    InvalidateCachedNzCount();
}

// -> dense
template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::DiagonalToDense() const
{
    int m = (int) GetNumRows();
    int n = (int) GetNumCols();

    if (m != n)
        LogicError("Diagonal can be called only for square matrix. (rows=%d, cols=%d)", m, n);

    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    GPUMatrix<ElemType> tmp(m, n, GetComputeDeviceId());

    // TODO: Implement optimized diagonal functions for sparse matrices. For now copy to dense first.
    CopyToDenseMatrix(tmp);

    return tmp.Diagonal();
}

// -> scalar
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::SumOfAbsElements() const
{
    if (IsEmpty())
        return 0;

    hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float res = 0;
        hipblasSasum(cuHandle, (int) GetNumNZElements(), reinterpret_cast<const float*>(NzValues()), 1, &res);
        return res;
    }
    else
    {
        double res = 0;
        hipblasDasum(cuHandle, (int) GetNumNZElements(), reinterpret_cast<const double*>(NzValues()), 1, &res);
        return ElemType(res);
    }
}

// -> scalar
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::SumOfElements() const
{
    if (IsEmpty())
        LogicError("SumOfElements: Matrix is empty");

    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);
    ElemType h_sum;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum1024Threads<ElemType><<<1, 1024>>>(NzValues(), d_sum, (LONG64) GetNumNZElements());
    CUDA_CALL(hipMemcpy(&h_sum, d_sum, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_sum);

    return h_sum;
}

// sqrt(sum all elements^2) -> scalar
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::FrobeniusNorm() const
{
    if (IsEmpty())
        return 0;

    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);
    ElemType h_sum = 0;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum21024Threads<ElemType><<<1, 1024>>>(NzValues(), d_sum, (int) GetNumNZElements());
    CUDA_CALL(hipMemcpy(&h_sum, d_sum, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_sum);

    if (sizeof(ElemType) == sizeof(float))
        return (ElemType) sqrtf((float) h_sum);
    else
        return (ElemType) sqrt((double) h_sum);
}

// -> scalar
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::MatrixNormInf() const
{
    if (IsEmpty())
        return 0;

    ElemType* d_maxAbs = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);
    ElemType h_maxAbs = 0;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionMatrixNormInf1024Threads<ElemType><<<1, 1024>>>(NzValues(), d_maxAbs, (int) GetNumNZElements());
    CUDA_CALL(hipMemcpy(&h_maxAbs, d_maxAbs, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_maxAbs);

    if (sizeof(ElemType) == sizeof(float))
        return h_maxAbs;
    else
        return h_maxAbs;
}

// -> scalar
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::MatrixNorm1() const
{
    return SumOfAbsElements();
}

#pragma endregion Member BLAS Functions

#pragma region Other Functions

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::ElementInverse()
{
#if 1
    // Note: This makes no sense because sparse matrices are defined by having lots of zeroes.
    NOT_IMPLEMENTED;
#else
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (IsEmpty())
        LogicError("ElementInverse: Matrix is empty.");

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _elemInverse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), N);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementInverseOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because sparse matrices are defined by having lots of zeroes.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    SetValue(a);
    return ElementInverse();
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSigmoid()
{
#if 1
    // Note: This makes no sense because sigmoid(0) != 0.
    NOT_IMPLEMENTED;
#else
    performElementWiseFunction(ElementWiseOperator::opSigmoid, *this);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSigmoidOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because sigmoid(0) != 0.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opSigmoid, a);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLinearRectifierDerivative()
{
    performElementWiseFunction(ElementWiseOperator::opLinearRectifierDerivative, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLinearRectifierDerivativeOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opLinearRectifierDerivative, a);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTanh()
{
    performElementWiseFunction(ElementWiseOperator::opTanh, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTanhOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opTanh, a);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSqrt()
{
    performElementWiseFunction(ElementWiseOperator::opSqrt, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSqrtOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opSqrt, a);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceExp()
{
#if 1
    // Note: This makes no sense because exp(0) != 0.
    NOT_IMPLEMENTED;
#else
    performElementWiseFunction(ElementWiseOperator::opExp, *this);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignExpOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because exp(0) != 0.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opExp, a);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLog()
{
#if 1
    // Note: This makes no sense because log(0) != 0.
    NOT_IMPLEMENTED;
#else
    performElementWiseFunction(ElementWiseOperator::opLog, *this);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLogOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because log(0) != 0.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opLog, a);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceAbs()
{
    performElementWiseFunction(ElementWiseOperator::opAbs, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignAbsOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        RequireSizeAndAllocate(a.GetNumRows(), a.GetNumCols(), a.NzCount());
    performElementWiseFunction(ElementWiseOperator::opAbs, a);
    return *this;
}

// TODO: Check whether these functions always map 0 to 0.
template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateBottom(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (IsEmpty())
        LogicError("InplaceTruncateBottom: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateBottom<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateBottomOf(const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (a.IsEmpty())
        LogicError("AssignTruncateBottomOf: Matrix a is empty.");

    if (this != &a)
    {
        // RequireSize(a.GetNumRows(), a.GetNumCols());
        ResizeAsAndCopyIndexFrom(a);
    }
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateBottom<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), a.NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateTop(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (IsEmpty())
        LogicError("InplaceTruncateTop: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateTop<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignOneHot(const GPUMatrix<ElemType>& a, vector<size_t>& shape, size_t axis)
{
    if (a.IsEmpty())
        LogicError("AssignOneHot: Matrix a is empty."); // BUGBUG: Just handle this gracefully.

    if (GetFormat() != matrixFormatSparseCSC)
        LogicError("AssignOneHot: Matrix format is not supported.");

    if (axis >= shape.size())
        LogicError("AssignOneHot: axis is not correct");

    int item_size = 1;
    for (size_t i = 0; i < shape.size() && i < axis; i++)
        item_size *= (int)shape[i];

    int num_class = (int)shape[axis];

    auto nRows = item_size * num_class;
    auto nCols = a.GetNumElements() / item_size;
    if (((GetNumRows() != 0) && (GetNumRows() != nRows)) || ((GetNumCols() != 0) && (GetNumCols() != nCols)))
        LogicError("AssignOneHot: Target matrix size is not correct");

    this->RequireSizeAndAllocate(nRows, nCols, a.GetNumElements());
    this->PrepareDevice();

    ElemType* indices = a.Data();
    GPUSPARSE_INDEX_TYPE* secondaryIndices = SecondaryIndexLocation();
    GPUSPARSE_INDEX_TYPE* majorIndices = MajorIndexLocation();
    ElemType* targetData = NzValues();
    CUDA_LONG N = (CUDA_LONG)a.GetNumElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignOneHotAsSparse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(indices, 
                                                                                    secondaryIndices,
                                                                                    majorIndices,
                                                                                    targetData,
                                                                                    num_class,
                                                                                    item_size,
                                                                                    N);

    UpdateCachedNzCount(a.GetNumElements());
    return *this;
}

// determines the row index of the row with the largest value in a column
template<class ElemType>
__global__ void _assignCSCArgmaxTo(ElemType *outData, CUDA_LONG numCols,
    const ElemType* nzValues,                 // base of nz-value array
    const GPUSPARSE_INDEX_TYPE* nzRowIndices, // base of corresponding row-index array
    const GPUSPARSE_INDEX_TYPE* colOffsets)   // array of offsets into nz array (including slice-view offset)
{
    // each thread processes one column, in a serial loop which is fine since this is meant for use with one-hot data
    const CUDA_LONG j = blockIdx.x;      // index of the column to process
    auto beginNZIndex = colOffsets[j];   // nz elements of this columns have this index range in the nz arrays
    auto endNZIndex   = colOffsets[j+1];
    ElemType bestVal = 0;   // (dummy)
    auto bestRowIndex = -1; // result for empty rows
    for (auto nzIndex = beginNZIndex; nzIndex != endNZIndex; nzIndex++)
    {
        if (bestRowIndex == -1 || bestVal < nzValues[nzIndex])
        {
            bestVal      = nzValues[nzIndex];
            bestRowIndex = nzRowIndices[nzIndex];
        }
    }
    outData[j] = (ElemType)bestRowIndex;
}

template <class ElemType>
/*static*/ void GPUSparseMatrix<ElemType>::AssignColumnwiseArgmaxTo(GPUMatrix<ElemType>& lhs, const GPUSparseMatrix<ElemType>& rhs)
{
    if (rhs.GetFormat() != matrixFormatSparseCSC)
        LogicError("AssignColumnwiseHardmaxTo: Argument must be in CSC format.");

    // output is a row vector
    let numCols = rhs.GetNumCols();
    lhs.Resize(1, numCols);

    // one thread per column (it's simple enough)
    SyncGuard syncGuard;
    if (numCols > 0)
        _assignCSCArgmaxTo<ElemType> <<<numCols, 1, 0, t_stream>>> (
            lhs.Data(), lhs.GetNumCols(), // target
            rhs.Buffer(),       // [nzIndex] rhs nz-element array base, without potential slice-view offset.
            rhs.RowLocation(),  // [nzIndex] rhs index array base, without potential slice-view offset.
            rhs.ColLocation()); // [colIndex] first nzIndex for rhs given column, with potential slice-view offset. End nzIndex is that of the next column.
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateTopOf(const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (a.IsEmpty())
        LogicError("AssignTruncateTopOf: Matrix a is empty.");

    if (this != &a)
    {
        ResizeAsAndCopyIndexFrom(a);
    }

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateTop<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), a.NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::SetToZeroIfAbsLessThan(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (IsEmpty())
        LogicError("SetToZeroIfAbsLessThan: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _setToZeroIfAbsLessThan<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), threshold, N);
    return *this;
}

#pragma endregion

#pragma region Helper Functions

// This is a memcpy() with built-in type cast.
// outBuffer should be allocated to be >= size by the caller
template <class ElemType>
template <class OutType, class InType>
/*private*/ void GPUSparseMatrix<ElemType>::ConvertBuffer(OutType* outBuffer, const InType* inBuffer, const size_t size)
{
#pragma omp parallel for
    for (size_t i = 0; i < (size & ~3); i += 4)
    {
        outBuffer[i] = inBuffer[i];
        outBuffer[i + 1] = inBuffer[i + 1];
        outBuffer[i + 2] = inBuffer[i + 2];
        outBuffer[i + 3] = inBuffer[i + 3];
    }
    // handle remaining stuffs
    for (size_t i = size & ~3; i < size; i++)
    {
        outBuffer[i] = inBuffer[i];
    }
}

template <class ElemType>
void* GPUSparseMatrix<ElemType>::ReserveTempHostBuffer(const size_t sizeInByte) const
{
    if (GetTempHostBufferSize() < sizeInByte)
    {
        delete[](byte*) GetTempHostBuffer();
        SetTempHostBuffer(new byte[sizeInByte]);
        SetTempHostBufferSize(sizeInByte);
    }
    return (void*) GetTempHostBuffer();
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::performElementWiseFunction(ElementWiseOperator kind, const GPUSparseMatrix<ElemType>& src)
{
    VerifyWritable(__FUNCTION__);

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    switch (kind)
    {
    case ElementWiseOperator::opSigmoid:
        return _elementWiseSigmoidOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opTanh:
        return _elementWiseTanhOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opSqrt:
        return _elementWiseSqrtOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opExp:
        return _elementWiseExpOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opLog:
        return _elementWiseLogOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opAbs:
        return _elementWiseAbsOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opLinearRectifierDerivative:
        return _elementWiseLinRectDerivativeOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    default:
        NOT_IMPLEMENTED;
    }
}

#pragma endregion Helper Functions

template class MATH_API GPUSparseMatrix<float>;
template class MATH_API GPUSparseMatrix<double>;

// We use Matrix<char> as the backing store for QuantizedMatrix
// Let's explicitly instantiate the methods we need for that purpose
template GPUSparseMatrix<char>::GPUSparseMatrix(DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<char>::GPUSparseMatrix(const size_t, const size_t, const size_t, DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<char>::GPUSparseMatrix(GPUSparseMatrix<char> const&);
template GPUSparseMatrix<char>::GPUSparseMatrix(GPUSparseMatrix<char>&&);
template void GPUSparseMatrix<char>::SetValue(CPUSparseMatrix<char> const&);
template void GPUSparseMatrix<char>::SetValue(GPUSparseMatrix<char> const&);
template void GPUSparseMatrix<char>::SetValue(GPUMatrix<char> const&);
//template void GPUSparseMatrix<char>::SetValue(CPUMatrix<char> const&);
template GPUMatrix<char> GPUSparseMatrix<char>::CopyToDenseMatrix() const;
template void GPUSparseMatrix<char>::CopyToDenseMatrix(GPUMatrix<char>&) const;
template void GPUSparseMatrix<char>::CopyToCPUSparseMatrix(CPUSparseMatrix<char>&) const;
template void GPUSparseMatrix<char>::ChangeDeviceTo(int);
template void GPUSparseMatrix<char>::Resize(const size_t, const size_t, const size_t, const bool);
template void GPUSparseMatrix<char>::RequireSizeAndAllocate(const size_t, const size_t, const size_t, const bool, const bool);
template void GPUSparseMatrix<char>::Reset();
template GPUSPARSE_INDEX_TYPE GPUSparseMatrix<char>::SecondaryIndexValueAt(size_t) const;
template GPUSparseMatrix<char>::~GPUSparseMatrix();
template GPUSparseMatrix<char> GPUSparseMatrix<char>::ColumnSlice(size_t, size_t) const;
template void GPUSparseMatrix<char>::AssignColumnSliceToDense(GPUMatrix<char>&, size_t, size_t) const;
template GPUMatrix<char> GPUSparseMatrix<char>::CopyColumnSliceToDense(size_t, size_t) const;
template GPUSparseMatrix<char>& GPUSparseMatrix<char>::operator=(GPUSparseMatrix<char>&&);
template void GPUSparseMatrix<char>::Reshape(const size_t, const size_t);
template void GPUSparseMatrix<char>::ScaleAndAdd(char, GPUSparseMatrix<char> const &, GPUMatrix<char> &);
template void GPUSparseMatrix<char>::ColumnwiseScaleAndWeightedAdd(char, const GPUSparseMatrix<char>&, const GPUMatrix<char>&, char, GPUMatrix<char>&);

// Support <short>
template GPUSparseMatrix<short>::GPUSparseMatrix(DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<short>::GPUSparseMatrix(const size_t, const size_t, const size_t, DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<short>::GPUSparseMatrix(GPUSparseMatrix<short> const&);
template GPUSparseMatrix<short>::GPUSparseMatrix(GPUSparseMatrix<short>&&);
template void GPUSparseMatrix<short>::SetValue(CPUSparseMatrix<short> const&);
template void GPUSparseMatrix<short>::SetValue(GPUSparseMatrix<short> const&);
template void GPUSparseMatrix<short>::SetValue(GPUMatrix<short> const&);
//template void GPUSparseMatrix<short>::SetValue(CPUMatrix<short> const&);
template GPUMatrix<short> GPUSparseMatrix<short>::CopyToDenseMatrix() const;
template void GPUSparseMatrix<short>::CopyToDenseMatrix(GPUMatrix<short>&) const;
template void GPUSparseMatrix<short>::CopyToCPUSparseMatrix(CPUSparseMatrix<short>&) const;
template void GPUSparseMatrix<short>::ChangeDeviceTo(int);
template void GPUSparseMatrix<short>::Resize(const size_t, const size_t, const size_t, const bool);
template void GPUSparseMatrix<short>::RequireSizeAndAllocate(const size_t, const size_t, const size_t, const bool, const bool);
template void GPUSparseMatrix<short>::Reset();
template GPUSPARSE_INDEX_TYPE GPUSparseMatrix<short>::SecondaryIndexValueAt(size_t) const;
template GPUSparseMatrix<short>::~GPUSparseMatrix();
template GPUSparseMatrix<short> GPUSparseMatrix<short>::ColumnSlice(size_t, size_t) const;
template void GPUSparseMatrix<short>::AssignColumnSliceToDense(GPUMatrix<short>&, size_t, size_t) const;
template GPUMatrix<short> GPUSparseMatrix<short>::CopyColumnSliceToDense(size_t, size_t) const;
template GPUSparseMatrix<short>& GPUSparseMatrix<short>::operator=(GPUSparseMatrix<short>&&);
template void GPUSparseMatrix<short>::Reshape(const size_t, const size_t);
template void GPUSparseMatrix<short>::ScaleAndAdd(short, GPUSparseMatrix<short> const &, GPUMatrix<short> &);
template void GPUSparseMatrix<short>::ColumnwiseScaleAndWeightedAdd(short, const GPUSparseMatrix<short>&, const GPUMatrix<short>&, short, GPUMatrix<short>&);

// Support <int>
template GPUSparseMatrix<int>::GPUSparseMatrix(DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<int>::~GPUSparseMatrix();
template void GPUSparseMatrix<int>::RequireSizeAndAllocate(const size_t, const size_t, const size_t, const bool, const bool);
template void GPUSparseMatrix<int>::Reset();

template <class ElemType>
MATH_API File& operator>>(File& stream, GPUSparseMatrix<ElemType>& us)
{
    us.VerifyWritable(__FUNCTION__);

    stream.GetMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
    size_t elsize;
    stream >> elsize;
    if (sizeof(ElemType) != elsize)
        RuntimeError("Template argument size doesn't match those in file");
    std::wstring matrixName;

    // now prepare this header to receive the data being read
    size_t nz, colnum, rownum;
    int format;

    // read in the header information
    stream >> matrixName >> format >> nz >> colnum >> rownum;

    us.SetFormat((MatrixFormat) format);
    if (us.GetFormat() != matrixFormatSparseCSC && us.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    us.RequireSizeAndAllocate(rownum, colnum, nz, true, false);

    if (nz > 0)
    {
        size_t compressedSize = (us.GetFormat() == matrixFormatSparseCSC) ? colnum + 1 : rownum + 1;
        ElemType* dataBuffer = new ElemType[nz];
        CPUSPARSE_INDEX_TYPE* unCompressedIndex = new CPUSPARSE_INDEX_TYPE[nz];
        CPUSPARSE_INDEX_TYPE* compressedIndex = new CPUSPARSE_INDEX_TYPE[compressedSize];

        // read in the sparse matrix info
        for (size_t i = 0; i < nz; ++i)
        {
            stream >> dataBuffer[i];
        }
        for (size_t i = 0; i < nz; ++i)
        {
            size_t val;
            stream >> val;
            unCompressedIndex[i] = val;
        }
        for (size_t i = 0; i < compressedSize; ++i)
        {
            size_t val;
            stream >> val;
            compressedIndex[i] = val;
        }

        if (us.GetFormat() == matrixFormatSparseCSC)
            us.SetMatrixFromCSCFormat(compressedIndex, unCompressedIndex, dataBuffer, nz, rownum, colnum);
        else if (us.GetFormat() == matrixFormatSparseCSR)
            us.SetMatrixFromCSRFormat(compressedIndex, unCompressedIndex, dataBuffer, nz, rownum, colnum);

        delete[] dataBuffer;
        delete[] unCompressedIndex;
        delete[] compressedIndex;
    }

    stream.GetMarker(fileMarkerEndSection, std::wstring(L"EMAT"));

    return stream;
}

template MATH_API File& operator>>(File& stream, GPUSparseMatrix<float>& us);
template MATH_API File& operator>>(File& stream, GPUSparseMatrix<double>& us);

template <class ElemType>
MATH_API File& operator<<(File& stream, const GPUSparseMatrix<ElemType>& us)
{
    if (us.GetFormat() != matrixFormatSparseCSC && us.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    stream.PutMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
    stream << sizeof(ElemType);
    std::wstring s(L"nnmatrix");
    stream << s;

    size_t nz = us.GetNumNZElements(), numElemAllocated = us.GetNumElemAllocated(), numRows = us.GetNumRows(), numCols = us.GetNumCols();
    size_t compressedSize = us.SecondaryIndexCount();
    int format = us.GetFormat();

    stream << format << nz << numCols << numRows;

    if (nz > 0)
    {
        ElemType* dataBuffer = nullptr;
        CPUSPARSE_INDEX_TYPE* compressedIndex = nullptr;
        CPUSPARSE_INDEX_TYPE* unCompressedIndex = nullptr;

        if (us.GetFormat() == matrixFormatSparseCSC)
            us.GetMatrixFromCSCFormat(compressedIndex, unCompressedIndex, dataBuffer, numElemAllocated, nz, numRows, numCols);
        else if (us.GetFormat() == matrixFormatSparseCSR)
            us.GetMatrixFromCSRFormat(compressedIndex, unCompressedIndex, dataBuffer, numElemAllocated, nz, numRows, numCols);
        else
            NOT_IMPLEMENTED;

        for (size_t i = 0; i < nz; ++i)
        {
            stream << dataBuffer[i];
        }
        for (size_t i = 0; i < nz; ++i)
        {
            size_t val = unCompressedIndex[i];
            stream << val;
        }
        for (size_t i = 0; i < compressedSize; ++i)
        {
            size_t val = compressedIndex[i];
            stream << val;
        }

        delete[] dataBuffer;
        delete[] unCompressedIndex;
        delete[] compressedIndex;
    }

    stream.PutMarker(fileMarkerEndSection, std::wstring(L"EMAT"));

    return stream;
}

template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<float>& us);
template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<double>& us);

}}}

#endif // CPUONLY
