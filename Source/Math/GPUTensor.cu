#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "Basics.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUTensor.h"
#include "GPUMatrix.h"
#include "GPUMatrixCUDAKernels.cuh"
#include "CommonMatrix.h"
#define TENSOR_OPS_DECL __device__ __host__
#include "TensorOps.h"
#include "fast_divmod.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <assert.h>
#include <limits.h>

// use fast divisor
#define USE_FAST_DIVMOD

#ifndef let
#define let const auto
#endif

#pragma comment(lib, "cudart.lib") // instruct linker to reference these libs
#pragma comment(lib, "cublas.lib")

#pragma warning(disable : 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning(disable : 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this
#pragma warning(disable : 4702) // unreachable code; triggered for unknown reasons

#ifdef _WIN32
// thread local storage to access the current stream, initalize to default stream
__declspec(thread)
#endif
extern hipStream_t t_stream;

namespace Microsoft { namespace MSR { namespace CNTK {

// =======================================================================
// TensorView support
// =======================================================================

// TensorView computes element-wise tensor operations.
//  - supports general strides
//  - input broadcasting is supported by stride=0
//  - the operation is denoted by an opCode
//  - reduction is supported, including summation, min, max (dual to broadcasting when computing gradients)
//  - reduction operation is given by an opCode: opSum, opMin, opMax and opLogSum.
//
// This library makes extensive use of templates and macros.
// Specifically, templates are used recursively to recurse over tensor dimensions.
// For example, a tensor op of rank K is computed by looping over the last dimension
// and then calling the same function template recursively with K-1.
// Template specializations exist in order to:
//  - terminate recursion
//  - optimize for thread-parallel reduction where elements are consecutive in memory
//
// The general algorithm is very straight forward:
//
//     for all output dimensions [###]:                                 // TensorOp()
//         output[###] *= beta
//         for all reduction dimensions [***]:                          // TensorOpWithReduction()
//             output[###] += op(input1[###,***], input1[###,***], ...) * alpha
//
// Indices and dimensions used throughout this code:
//  - NUM_ARGS       = N = ariness+1; number of arguments *including output* (binary op: N=3)
//  - REGULAR_RANK   = K = rank of output elements, regularOpDims.size(). K=0 means scalar.
//  - REGULAR_AXIS   = k = -1..K-1 = recursion index
//  - REDUCTION_RANK = M = reduction rank, reducingOpDims.size(). M=0 means no reduction.
//  - REGULAR_AXIS   = m = -1..M-1 = recursion index
//
// Other frequently used variable names:
//  - alpha, beta: BLAS-style weights: outVal = beta * outVal + alpha * f(inVals)
//                 where beta=0 is an assignment (0 * outVal := 0, even e.g. if outVal = NaN)
//  - pointers[N]:          pointer to first element, for each argument
//  - regularOpDims[K]:     tensor dimensions of output elements to produce
//  - regularStrides[N,K]:  strides; multiply index[k] with strides[n,k] to get element offset for this dimension
//                          Broadcasting of inputs is implemented by a stride being 0.
//  - reducingOpDims[M]:    tensor dimensions of input elements to reduce over
//  - reducingStrides[N,M]: strides for input reduction. Always 0 for output argument.
//
// This code uses two custom structs, FixedArray<> and FixedMatrix<>, which
// are templated equivalents to vector<> and vector<vector<>> for CUDA code.

// -----------------------------------------------------------------------
// simple fixed-size arrays for passing dimension information by value
// since CUDA can't just take our std::array and std::vector
// -----------------------------------------------------------------------

template <typename T, size_t N>
struct FixedArray
{
    T m_data[N];
    __device__ __host__ size_t size() const { return N; }
    __device__ __host__ T& operator[](size_t n) { return m_data[n]; }
    __device__ __host__ T operator[](size_t n) const { return m_data[n]; }
    template <class VEC>
    FixedArray(const VEC& data) // construct from CPU-side STL array or vector
    {
        assert(data.size() == N);
        for (size_t n = 0; n < N; n++)
        {
            m_data[n] = (T) data[n];
            if (m_data[n] != data[n]) // overflow check
                InvalidArgument("FixedArray: Dimensions out of range, too few bits.");
        }
    }
};
template <typename T> // specialized version for 0 elements
struct FixedArray<T, 0>
{
    __device__ __host__ size_t size() const { return 0; }
    template <class VEC>
    FixedArray(const VEC& data)
    {
        assert(data.size() == 0);
        UNUSED(data);
    }
    FixedArray() { }
};

template <typename T, size_t N, size_t K> // N = which input/output; K = index depth
struct FixedMatrix
{
    T m_data[N][K];
    __device__ __host__ size_t getNumRows() const { return N; }
    __device__ __host__ size_t getNumCols() const { return K; }
    __device__ __host__ T& operator()(size_t n, size_t k) { return m_data[n][k]; }
    __device__ __host__ T operator()(size_t n, size_t k) const { return m_data[n][k]; }
    template <typename U>
    FixedMatrix(const array<SmallVector<U>, N>& data) // construct from CPU-side array of vectors
    {
        assert(data.size() == N);
        for (size_t n = 0; n < N; n++)
        {
            assert(data[n].size() == K);
            for (size_t k = 0; k < K; k++)
            {
                m_data[n][k] = (T) data[n][k];
                if (m_data[n][k] != data[n][k]) // overflow check
                    InvalidArgument("FixedArray: Dimensions out of range, too few bits.");
            }
        }
    }
};
template <typename T, size_t N> // specialized version for 0 elements
struct FixedMatrix<T, N, 0>
{
    __device__ __host__ size_t getNumRows() const { return N; }
    __device__ __host__ size_t getNumCols() const { return 0; }
    template <typename U>
    FixedMatrix(const array<SmallVector<U>, N>& data)
    {
        assert(data.size() == N);
        for (size_t n = 0; n < N; n++)
            assert(data[n].size() == 0);
        UNUSED(data);
    }
    FixedMatrix() { }
};

// -----------------------------------------------------------------------
// function to actually compute a function of (N-1) inputs based on the opcode
// -----------------------------------------------------------------------

//template <class ElemType>
//struct TensorOps
//{
    template <class ElemType>
    static __device__ ElemType Op(const FixedArray<ElemType*, /*NUM_ARGS=*/1>& pointers, ElementWiseOperator op)
    {
#define CaseNullaryTensorOp(oper)       \
    case ElementWiseOperator::op##oper: \
        return Op##oper<ElemType>()
        switch (op)
        {
            ForAllNullaryOps(CaseNullaryTensorOp);
        default:
            return OpConstOne<ElemType>(); // (failure--we only have one nullary op, so use the same, maybe it will eliminate the switch altogether)
        }
    }
    template <class ElemType>
    static __device__ ElemType Op(const FixedArray<ElemType*, /*NUM_ARGS=*/2>& pointers, ElementWiseOperator op)
    {
        ElemType a = *(pointers[0]);
#define CaseUnaryTensorOp(oper)         \
    case ElementWiseOperator::op##oper: \
        return Op##oper(a)
        switch (op)
        {
            ForAllUnaryOps(CaseUnaryTensorOp);
        default:
            return 0; // (failure)
        }
    }
    template <class ElemType>
    static __device__ ElemType Op(const FixedArray<ElemType*, /*NUM_ARGS=*/3>& pointers, ElementWiseOperator op)
    {
        // const ElemType & a = *(pointers[0]);    // const & for opIndex--costs quite some code bloat
        ElemType a = *(pointers[0]);
        ElemType b = *(pointers[1]);
#define CaseBinaryTensorOp(oper)        \
    case ElementWiseOperator::op##oper: \
        return Op##oper(a, b)
        switch (op)
        {
            ForAllBinaryOps(CaseBinaryTensorOp); // note: this costs about 6% compared to having only a single case
        default:
            return 0; // (failure)
        }
    }
    template <class ElemType>
    static __device__ ElemType Op(const FixedArray<ElemType*, /*NUM_ARGS=*/4>& pointers, ElementWiseOperator op)
    {
#define CaseTernaryTensorOp(oper)       \
    case ElementWiseOperator::op##oper: \
        return Op##oper(*(pointers[0]), *(pointers[1]), *(pointers[2])) // reading each time, which saves mem accesses for OpCond
        switch (op)
        {
            ForAllTernaryOps(CaseTernaryTensorOp);
        default:
            return 0; // (failure)
        }
    }
    template <class ElemType>
    static __device__ ElemType Op(const FixedArray<ElemType*, /*NUM_ARGS=*/5>& pointers, ElementWiseOperator op)
    {
#define CaseQuaternaryTensorOp(oper)       \
    case ElementWiseOperator::op##oper: \
        return Op##oper(*(pointers[0]), *(pointers[1]), *(pointers[2]), *(pointers[3])) // reading each time, which saves mem accesses for OpCond
        switch (op)
        {
            ForAllQuaternaryOps(CaseQuaternaryTensorOp);
        default:
            return 0; // (failure)
        }
    }
//};

// ----------------------------------------------------------------------------
// Function to update an aggregate value for the specified reduction operation
// ----------------------------------------------------------------------------

template <typename ElemType> __device__ ElemType AggregateNeutralValue(ElementWiseOperator op)
{
    return 0; // error, only the explicit instantiations below should be used.
};

template<> __device__ float AggregateNeutralValue<float>(ElementWiseOperator op)
{
    switch (op)
    {
    case ElementWiseOperator::opSum:                return 0;
    case ElementWiseOperator::opLogSum:             return -FLT_MAX; // note: do not use INFINITY anywhere here, as it causes NaNs
    case ElementWiseOperator::opMin:                return FLT_MAX;
    case ElementWiseOperator::opMax:                return -FLT_MAX;
    case ElementWiseOperator::opElementwiseProduct: return 1.0f;
    case ElementWiseOperator::opArgmin:             return FLT_MAX;
    case ElementWiseOperator::opArgmax:             return -FLT_MAX;
    default:                                        return 0; // error
    }
};

template<> __device__ double AggregateNeutralValue<double>(ElementWiseOperator op)
{
    switch (op)
    {
    case ElementWiseOperator::opSum:                return 0;
    case ElementWiseOperator::opLogSum:             return -DBL_MAX;
    case ElementWiseOperator::opMin:                return DBL_MAX;
    case ElementWiseOperator::opMax:                return -DBL_MAX;
    case ElementWiseOperator::opElementwiseProduct: return 1.0;
    case ElementWiseOperator::opArgmin:             return DBL_MAX;
    case ElementWiseOperator::opArgmax:             return -DBL_MAX;
    default:                                        return 0; // error
    }
};


template<typename ReductionType, class ElemType> __device__ void Aggregate(ReductionType& aggregate, ElemType val, ElementWiseOperator reductionOp)
{
    switch (reductionOp)
    {
    case ElementWiseOperator::opSum:                aggregate += val;                     break;
    case ElementWiseOperator::opLogSum:             aggregate = OpLogSum(aggregate, val); break;
    case ElementWiseOperator::opElementwiseProduct: aggregate *= val;                     break;
    case ElementWiseOperator::opMin:                if (val < aggregate) aggregate = val; break;
    case ElementWiseOperator::opMax:                if (val > aggregate) aggregate = val; break;
    }
};

// -----------------------------------------------------------------------
// function to compute the value for a given output location (including reduction)
// -----------------------------------------------------------------------

#define ReduceElemType ElemType // (note: we could use 'double' here, but that would cause problems with CUDA cards that don't support double)

template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int m>
struct TensorOpReduce
{
    // this version for m >= 0
    static __device__ ElemType Compute(FixedArray<ElemType*, NUM_ARGS> pointers,
                                       ElementWiseOperator op, ElementWiseOperator reductionOp,
                                       const FixedArray<C_unsigned_int, REDUCTION_RANK>& reducingOpDims, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& reducingStrides)
    {
        // start with index 0
        // We may use 'double' since we are memory-bound anyway.
        ReduceElemType aggregate = TensorOpReduce<ElemType, NUM_ARGS, REDUCTION_RANK, m - 1>::Compute(pointers, op, reductionOp, reducingOpDims, reducingStrides);
        // apply this index to the pointers
        C_size_t dim = reducingOpDims[m];
        for (C_size_t k = 1 /*done with k=0 already*/; k < dim; k++)
        {
            // bump the pointers
            #pragma unroll
            for (C_size_t i = 0; i < NUM_ARGS - 1; i++) // NUM_ARGS-1 because output is not used here
            {
                pointers[i] += reducingStrides(i, (C_size_t) m);
            }
            ElemType val = TensorOpReduce<ElemType, NUM_ARGS, REDUCTION_RANK, m - 1>::Compute(pointers, op, reductionOp, reducingOpDims, reducingStrides);
            Aggregate<ReduceElemType, ElemType>(aggregate, val, reductionOp);
        }
        return (ElemType) aggregate;
    }
};

// this one terminates the template recursion over reduction dimensions
// The pointers are pointing to the input element.
template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK>
struct TensorOpReduce<ElemType, NUM_ARGS, REDUCTION_RANK, /*m=*/-1>
{
    // this version for m = -1
    // the pointers are pointing to the right location(s) to take the operation over
    static __device__ ElemType Compute(FixedArray<ElemType*, NUM_ARGS> pointers,
                                       ElementWiseOperator op, ElementWiseOperator reductionOp,
                                       const FixedArray<C_unsigned_int, REDUCTION_RANK>& /*reducingOpDims*/, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& /*reducingStrides*/)
    {
        return Op(pointers, op); // finally computing something!
    }
};

// Similar to TensorOpReduce but count the number of elements seen so far and keep track
// of the index of the last element assigned to the aggregate. It assume that reduction is done
// in a single thread.
template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int m>
struct TensorArgOpReduce
{
    // this version for m >= 0
    static __device__ ElemType Compute(FixedArray<ElemType*, NUM_ARGS> pointers,
                                       ElementWiseOperator reductionOp,
                                       const FixedArray<C_unsigned_int, REDUCTION_RANK>& reducingOpDims, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& reducingStrides, 
                                       C_unsigned_int& count, C_unsigned_int& index)
    {
        // start with index 0
        ReduceElemType aggregate = TensorArgOpReduce<ElemType, NUM_ARGS, REDUCTION_RANK, m - 1>::Compute(pointers, reductionOp, reducingOpDims, reducingStrides, count, index);
        // apply this index to the pointers
        C_size_t dim = reducingOpDims[m];
        for (C_size_t k = 1 /*done with k=0 already*/; k < dim; k++)
        {
            // bump the pointers
#pragma unroll
            for (C_size_t i = 0; i < NUM_ARGS - 1; i++) // NUM_ARGS-1 because output is not used here
            {
                pointers[i] += reducingStrides(i, (C_size_t)m);
            }

            ElemType val = TensorArgOpReduce<ElemType, NUM_ARGS, REDUCTION_RANK, m - 1>::Compute(pointers, reductionOp, reducingOpDims, reducingStrides, count, index);
            bool update = false;
            switch (reductionOp)
            {
                case ElementWiseOperator::opArgmin:
                    update = (aggregate > val);
                    break;
                case ElementWiseOperator::opArgmax:
                    update = (aggregate < val);
                    break;
            }

            if (update)
            {
                aggregate = val;
                index = count - 1;
            }
        }
        return (ElemType)aggregate;
    }
};

// this one terminates the template recursion over reduction dimensions
// The pointers are pointing to the input element.
template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK>
struct TensorArgOpReduce<ElemType, NUM_ARGS, REDUCTION_RANK, /*m=*/-1>
{
    // this version for m = -1
    // the pointers are pointing to the right location(s) to take the operation over
    static __device__ ElemType Compute(FixedArray<ElemType*, NUM_ARGS> pointers,
                                       ElementWiseOperator reductionOp,
                                       const FixedArray<C_unsigned_int, REDUCTION_RANK>& /*reducingOpDims*/, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& /*reducingStrides*/,
                                       C_unsigned_int& count, C_unsigned_int& index)
    {
        count++;
        return *(pointers[0]);
    }
};

// -----------------------------------------------------------------------
// function to compute one constituent of the value for a given output location
// (reduction is not done here, but by calling into here multiple times)
// -----------------------------------------------------------------------

template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int m>
struct TensorOpParallelReduce
{
    // this version for m >= 0
    static __device__ ElemType Compute(CUDA_LONG id, FixedArray<ElemType*, NUM_ARGS> pointers,
                                       ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, REDUCTION_RANK>& reducingOpDims, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& reducingStrides,
                                       FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
    {
        // map id (location on grid) to index[k]
        C_size_t stride = 1; // compute the stride. This seems expensive, but since we we only currently support REDUCTION_RANK <= 2, this is just compile-time selection between 1 and reducingOpDims[0].
        #pragma unroll
        for (int i = 0; i < m; i++)
        {
            stride *= reducingOpDims[(C_size_t) i];
        }

        C_size_t index;
#ifndef USE_FAST_DIVMOD
        index = id / stride; // this dimension. For m=0, the stride is 1 and hence the division will be removed at compile time.
        // id = id % stride;             // remaining dimensions inside this. For m=0 this value is ignored and hence not even computed.
        id = id - stride*index;             // remaining dimensions inside this. For m=0 this value is ignored and hence not even computed.
#else
        if (m == 0)
        {
            index = id;
            id = 0;
        }
        else
        {
            reducingOpDimDivmod[m].divmod(id, index, id);
        }
#endif
        // apply this index to the pointers
        #pragma unroll
        for (C_size_t i = 0; i < NUM_ARGS - 1; i++)
        {
            pointers[i] += index * reducingStrides(i, (C_size_t) m); // now this dimension is taken care of
        }
        return TensorOpParallelReduce<ElemType, NUM_ARGS, REDUCTION_RANK, m - 1>::Compute(id, pointers, op, reducingOpDims, reducingStrides, reducingOpDimDivmod);
    }
};

// this one terminates the template recursion over reduction dimensions
// The pointers are pointing to the input element.
template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK>
struct TensorOpParallelReduce<ElemType, NUM_ARGS, REDUCTION_RANK, /*m=*/-1>
{
    // this version for m = -1
    // the pointers are pointing to the right location(s) to take the operation over
    static __device__ ElemType Compute(CUDA_LONG /*id*/, FixedArray<ElemType*, NUM_ARGS> pointers,
                                       ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, REDUCTION_RANK>& /*reducingOpDims*/, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& /*reducingStrides*/,
                                       FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
    {
        return Op(pointers, op); // finally computing something!
    }
};

// -----------------------------------------------------------------------
// perform loop over regular index k for NUM_ARGS-nary operations (NUM_ARGS counting the output)
// -----------------------------------------------------------------------

// The 'pointers' only refer to a single element, so we will bump them in-place to perform indexing.
template <C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK, bool PARALLEL_REDUCE, C_int k>
struct TensorOpElement
{
    // template-recursive version loops over indices
    template<class ElemType>
    static __device__ void Compute(CUDA_LONG id, ElemType beta, FixedArray<ElemType*, NUM_ARGS>& pointers,
                                   ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, REGULAR_RANK>& regularOpStrides, const FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK>& regularStrides,
                                   const FixedArray<C_unsigned_int, REDUCTION_RANK>& reducingOpDims, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& reducingStrides,
                                   CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize,
                                   FixedArray<fast_divmod, REGULAR_RANK> regularOpStrideDivmod, FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
    {
        // map id (location on grid) to index[k]
#ifndef USE_FAST_DIVMOD
        C_size_t stride = regularOpStrides[(C_size_t) k];
        C_size_t index = id / stride; // this dimension
        // id = id % stride;             // remaining dimensions inside this
        id = id - stride*index;             // remaining dimensions inside this
#else
        C_size_t index;
        regularOpStrideDivmod[k].divmod(id, index, id);
#endif
        // apply this index to the pointers
        #pragma unroll
        for (C_size_t i = 0; i < NUM_ARGS; i++) {
            pointers[i] += index * regularStrides(i, (C_size_t) k); // now this dimension is taken care of
        }
        // process the previous index
        TensorOpElement<NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, PARALLEL_REDUCE, k - 1>::Compute(
            id, beta, pointers,
            alpha, op, reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize,
            regularOpStrideDivmod, reducingOpDimDivmod);
    }
};

// specialization for k=0 where op stride is guaranteed to be 1
template <C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK, bool PARALLEL_REDUCE>
struct TensorOpElement<NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, PARALLEL_REDUCE, /*k=*/0>
{
    // template-recursive version loops over indices
    template<class ElemType>
    static __device__ void Compute(CUDA_LONG id, ElemType beta, FixedArray<ElemType*, NUM_ARGS>& pointers,
                                   ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, REGULAR_RANK>& regularOpStrides, const FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK>& regularStrides,
                                   const FixedArray<C_unsigned_int, REDUCTION_RANK>& reducingOpDims, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& reducingStrides,
                                   CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize,
                                   FixedArray<fast_divmod, REGULAR_RANK> regularOpStrideDivmod, FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
    {
        // map id (location on grid) to index[k]
        C_size_t index = id; // this dimension
        // apply this index to the pointers
        #pragma unroll
        for (C_size_t i = 0; i < NUM_ARGS; i++)
        {
            pointers[i] += index * regularStrides(i, 0); // now this dimension is taken care of
        }
        // process the previous index
        TensorOpElement<NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, PARALLEL_REDUCE, -1>::Compute(
            /*id*/ 0, beta, pointers,
            alpha, op, reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize,
            regularOpStrideDivmod, reducingOpDimDivmod);
    }
};

// specialization for k = -1 terminates the template recursion, and computes reductions in a for loop
template <C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK>
struct TensorOpElement<NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, /*PARALLEL_REDUCE=*/false, /*k=*/-1>
{
    // template-recursion-teminating version computes the actual value for this output location
    // now the output pointers point to the right element (input pointers may still iterate for reduction)
    template<class ElemType>
    static __device__ void Compute(CUDA_LONG /*id*/, ElemType beta, FixedArray<ElemType*, NUM_ARGS>& pointers,
                                   ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, REGULAR_RANK>& /*regularOpStrides*/, const FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK>& /*regularStrides*/,
                                   const FixedArray<C_unsigned_int, REDUCTION_RANK>& reducingOpDims, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& reducingStrides, CUDA_LONG /*reductionBegin*/, CUDA_LONG /*reductionChunkSize*/,
                                   FixedArray<fast_divmod, REGULAR_RANK> regularOpStrideDivmod, FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
    {
        // compute the operation for this output coordinate
        // This may still involve a reduction over inverse-broadcasting dimensions.
        ElemType val = TensorOpReduce<ElemType, NUM_ARGS, REDUCTION_RANK, REDUCTION_RANK - 1>::Compute(pointers, op, reductionOp, reducingOpDims, reducingStrides);
        // scale
        val *= alpha;
        // combine with previous value in target matrix, then write it out
        if (NUM_ARGS < 4 || val != 0 || beta != 1) // (skip memory access if not needed) (NUM_ARGS<4: skip this test)
        {
            auto* pout = pointers[pointers.size() - 1];
            if (beta != 0) // (skip memory access if not needed, and allow for ignoring NaNs)
                val += beta * *pout;
            // save
            *pout = val;
        }
    }
};

#undef ALLOW_ATOMIC_REDUCTION // undefine to disable use of atomicAdd() below, for testing it

// specialization for k = -1 terminates the template recursion, and computes reductions in parallel
template <C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK>
struct TensorOpElement<NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, /*PARALLEL_REDUCE=*/true, /*k=*/-1>
{
    // template-recursion-teminating version computes the actual value for this output location
    // now the output pointers point to the right element (input pointers may still iterate for reduction)
    template<class ElemType>
    static __device__ void Compute(CUDA_LONG /*id*/, ElemType beta, FixedArray<ElemType*, NUM_ARGS>& pointers,
                                   ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, REGULAR_RANK>& /*regularOpStrides*/, const FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK>& /*regularStrides*/,
                                   const FixedArray<C_unsigned_int, REDUCTION_RANK>& reducingOpDims, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& reducingStrides, CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize,
                                   FixedArray<fast_divmod, REGULAR_RANK> regularOpStrideDivmod, FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
    {
        CUDA_LONG reductionBlock = blockIdx.z; // reduction-block index  --larger reductions are split into blocks
        CUDA_LONG tid = threadIdx.x;           // thread index
        CUDA_LONG tids = blockDim.x;           // out of how many threads  --note: last block is partial

        // determine our range  --this is a single int mul, we can stomach it (we could alternatively pass in yet another parameter)
        CUDA_LONG reductionDim = (CUDA_LONG) reducingOpDims[0];
        for (C_size_t i = 1; i < reducingOpDims.size(); i++)
            reductionDim *= reducingOpDims[i];

        // determine the redId range that we operate on
        // Each thread takes a stride tid + (multiples of tids) within this range.
        reductionBegin += reductionChunkSize * reductionBlock;
        CUDA_LONG reductionEnd = min(reductionBegin + reductionChunkSize, reductionDim);

        // compute the operation for this input coordinate
        ReduceElemType aggregate = AggregateNeutralValue<ReduceElemType>(reductionOp);

        for (CUDA_LONG redId = reductionBegin + tid; redId < reductionEnd; redId += tids)
        {
            auto val = TensorOpParallelReduce<ElemType, NUM_ARGS, REDUCTION_RANK, REDUCTION_RANK - 1>::Compute(redId, pointers, op, reducingOpDims, reducingStrides, reducingOpDimDivmod);
            Aggregate<ReduceElemType, ElemType>(aggregate, val, reductionOp);
        }

        // reduce    --cf https://docs.nvidia.com/cuda/samples/6_Advanced/reduction/doc/reduction.pdf
        __shared__ ReduceElemType volatile accumulators[GridDim::maxThreadsPerBlock /*tids == blockDim.x, as specified at launch*/];
        accumulators[tid] = aggregate;
        __syncthreads();
        static_assert(GridDim::maxThreadsPerBlock <= 1024, "GridDim::maxThreadsPerBlock too large, need to add manually unrolled steps");
        for (CUDA_LONG i = 512; i; i >>= 1)
        {
            if (tid < i && tid + i < tids)
                Aggregate<volatile ReduceElemType, volatile ReduceElemType>(accumulators[tid], accumulators[tid + i], reductionOp);

            if (0 + i < tids)
                __syncthreads(); // sync if condition true for at least one thread
            // TODO: use volatile* and then we can skip the __syncthreads() for the last 32 values. See Amit's allreduce() function implementation in MatrixQuantizer_kernel.cu.
        }

        // now set final value to output coordinate
        if (tid == 0)
        {
            ElemType val = (ElemType) accumulators[0];
            // scale
            val *= alpha;
            // combine with previous value in target matrix, then write it out
            if (NUM_ARGS < 4 || val != 0 || beta != 1) // (skip memory access if not needed) (NUM_ARGS<4: skip this test)
            {
                auto* pout = pointers[pointers.size() - 1];
#ifdef ALLOW_ATOMIC_REDUCTION
                CUDA_LONG reductionBlocks = gridDim.z; // number of reduction blocks. If >1 we need atomicAdd
                if (reductionBlocks > 1) // multiple blocks: need to use atomicAdd()
                {
                    // in this case, outer calling code must pass beta = 1
                    atomicAdd(pout, val);
                }
                else
#endif
                {
                    if (beta != 0)
                        val += beta * *pout;
                    // save
                    *pout = val;
                }
            }
        }
    }
};

// -----------------------------------------------------------------------
// perform loop over regular index k for NUM_ARGS-nary operations (NUM_ARGS counting the output)
// keep track of the indices.
// -----------------------------------------------------------------------

// The 'pointers' only refer to a single element, so we will bump them in-place to perform indexing.
template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK, C_int k>
struct TensorArgOpElement
{
    // template-recursive version loops over indices
    static __device__ void Compute(CUDA_LONG id, FixedArray<ElemType*, NUM_ARGS>& pointers,
                                   ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, REGULAR_RANK>& regularOpStrides, const FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK>& regularStrides,
                                   const FixedArray<C_unsigned_int, REDUCTION_RANK>& reducingOpDims, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& reducingStrides,
                                   CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize,
                                   FixedArray<fast_divmod, REGULAR_RANK> regularOpStrideDivmod, FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
    {
        // map id (location on grid) to index[k]
#ifndef USE_FAST_DIVMOD
        C_size_t stride = regularOpStrides[(C_size_t)k];
        C_size_t index = id / stride; // this dimension
                                      // id = id % stride;             // remaining dimensions inside this
        id = id - stride*index;             // remaining dimensions inside this
#else
        C_size_t index;
        regularOpStrideDivmod[k].divmod(id, index, id);
#endif
        // apply this index to the pointers
#pragma unroll
        for (C_size_t i = 0; i < NUM_ARGS; i++) {
            pointers[i] += index * regularStrides(i, (C_size_t)k); // now this dimension is taken care of
        }
        // process the previous index
        TensorArgOpElement<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, k - 1>::Compute(id, pointers, reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize,
            regularOpStrideDivmod, reducingOpDimDivmod);
    }
};

// specialization for k = -1 terminates the template recursion, and computes reductions in a for loop
template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK>
struct TensorArgOpElement<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, /*k=*/-1>
{
    // template-recursion-teminating version computes the actual value for this output location
    // now the output pointers point to the right element (input pointers may still iterate for reduction)
    static __device__ void Compute(CUDA_LONG /*id*/, FixedArray<ElemType*, NUM_ARGS>& pointers,
                                   ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, REGULAR_RANK>& /*regularOpStrides*/, const FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK>& /*regularStrides*/,
                                   const FixedArray<C_unsigned_int, REDUCTION_RANK>& reducingOpDims, const FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK>& reducingStrides, CUDA_LONG /*reductionBegin*/, CUDA_LONG /*reductionChunkSize*/,
                                   FixedArray<fast_divmod, REGULAR_RANK> regularOpStrideDivmod, FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
    {
        // compute the operation for this output coordinate
        // This may still involve a reduction over inverse-broadcasting dimensions.
        C_unsigned_int count = 0;
        C_unsigned_int index = 0;
        ElemType val = TensorArgOpReduce<ElemType, NUM_ARGS, REDUCTION_RANK, REDUCTION_RANK - 1>::Compute(pointers, reductionOp, reducingOpDims, reducingStrides, count, index);

        // combine with previous value in target matrix, then write it out
        if (NUM_ARGS < 4 || val != 0) // (skip memory access if not needed) (NUM_ARGS<4: skip this test)
        {
            auto* pout = pointers[pointers.size() - 1];

            // save
            *pout = (ElemType) index;
        }
    }
};

// -----------------------------------------------------------------------
// kernel and launch  --no reduction
// -----------------------------------------------------------------------

// launch tensor op with CUDA
template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK>
__global__ void _launchTensorOp(ElemType beta, FixedArray<ElemType*, NUM_ARGS> pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                FixedArray<C_unsigned_int, REGULAR_RANK> regularOpStrides, FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK> regularStrides, CUDA_LONG numElements,
                                FixedArray<C_unsigned_int, REDUCTION_RANK> reducingOpDims, FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK> reducingStrides,
                                FixedArray<fast_divmod, REGULAR_RANK> regularOpStrideDivmod, FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id < numElements) // note: there are no __syncthread() calls inside
        TensorOpElement<NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, false, REGULAR_RANK - 1>::Compute(
            id, beta, pointers,
            alpha, op, reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, 0, 0,
            regularOpStrideDivmod, reducingOpDimDivmod);
}

template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK>
__global__ void _launchTensorArgOp(FixedArray<ElemType*, NUM_ARGS> pointers,
                                   ElementWiseOperator reductionOp,
                                   FixedArray<C_unsigned_int, REGULAR_RANK> regularOpStrides, FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK> regularStrides, CUDA_LONG numElements,
                                   FixedArray<C_unsigned_int, REDUCTION_RANK> reducingOpDims, FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK> reducingStrides,
                                   FixedArray<fast_divmod, REGULAR_RANK> regularOpStrideDivmod, FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id < numElements) // note: there are no __syncthread() calls inside
        TensorArgOpElement<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, REGULAR_RANK - 1>::Compute(
            id, pointers,
            reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, 0, 0,
            regularOpStrideDivmod, reducingOpDimDivmod);
}

template <class ElemType, C_size_t NUM_ARGS, C_int REGULAR_RANK>
static void LaunchTensorOp(ElemType beta, array<ElemType*, NUM_ARGS> pointerVector,
                           ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                           const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, NUM_ARGS>& regularStrideVectors)
{
    // copy all parameters to CUDA-compatible data structures
    FixedArray<ElemType*, NUM_ARGS> pointers(pointerVector);
    SmallVector<C_size_t> regularOpStrideVector; // kernel needs the strides for converting thread index back to multi-dimensional tensor index
    C_size_t numElements = 1;
    // input divisors
    SmallVector<fast_divmod> regularOpStrideDivmodVector;
    for (C_size_t k = 0; k < regularOpDims.size(); k++)
    {
        regularOpStrideVector.push_back(numElements);
        // create fast division objects
        regularOpStrideDivmodVector.push_back(fast_divmod(numElements));
        numElements *= (C_size_t) regularOpDims[k];
    }

    SmallVector<fast_divmod> reducingOpDimDivmodVector;

    FixedArray<C_unsigned_int,     REGULAR_RANK      > regularOpStrides(regularOpStrideVector);
    FixedMatrix<C_int, NUM_ARGS,   REGULAR_RANK      > regularStrides(regularStrideVectors);
    FixedArray<C_unsigned_int,   /*REDUCTION_RANK=*/0> reducingOpDims; // empty reduction dimensions
    FixedMatrix<C_int, NUM_ARGS, /*REDUCTION_RANK=*/0> reducingStrides;
    // reduced divisors
    FixedArray<fast_divmod,         REGULAR_RANK      > regularOpStrideDivmod(regularOpStrideDivmodVector);
    FixedArray<fast_divmod,       /*REDUCTION_RANK=*/0> reducingOpDimDivmod;

    // launch the kernel
    CUDA_LONG NN = (CUDA_LONG) numElements; // linear space identifying each individual input element
    SyncGuard syncGuard;
    GridDim grid(NN);
    if (reductionOp == ElementWiseOperator::opArgmax || reductionOp == ElementWiseOperator::opArgmin)
    {
        if (alpha != 1 || beta != 0 || op != opCopy)
            InvalidArgument("LaunchTensorOp: Argmin/max reductions require opCopy, alpha=1, and beta=0");
        _launchTensorArgOp<ElemType, NUM_ARGS, /*REDUCTION_RANK=*/0, REGULAR_RANK> << <grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream >> > (
            pointers,
            reductionOp,
            regularOpStrides, regularStrides, grid.m_N,
            reducingOpDims, reducingStrides,
            regularOpStrideDivmod, reducingOpDimDivmod);
    }
    else
    {
        _launchTensorOp<ElemType, NUM_ARGS, /*REDUCTION_RANK=*/0, REGULAR_RANK> << <grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream >> > (
            beta, pointers,
            alpha, op, (ElementWiseOperator)(-1) /* dummy reductionOp */, regularOpStrides, regularStrides,
            grid.m_N, reducingOpDims, reducingStrides,
            regularOpStrideDivmod, reducingOpDimDivmod);
    }
}

// -----------------------------------------------------------------------
// kernel and launch  --with reduction
// -----------------------------------------------------------------------

template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK>
__global__ void _launchTensorOpWithReduction(ElemType beta, FixedArray<ElemType*, NUM_ARGS> pointers,
                                             ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                             FixedArray<C_unsigned_int, REGULAR_RANK> regularOpStrides, FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK> regularStrides, CUDA_LONG numElements,
                                             FixedArray<C_unsigned_int, REDUCTION_RANK> reducingOpDims, FixedMatrix<C_int, NUM_ARGS, REDUCTION_RANK> reducingStrides,
                                             CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize,
                                             FixedArray<fast_divmod, REGULAR_RANK> regularOpStrideDivmod, FixedArray<fast_divmod, REDUCTION_RANK> reducingOpDimDivmod)
{
    CUDA_LONG id = gridDim.x * blockIdx.y + blockIdx.x; // input dimensions are Y dimension of blocks in this case, so we can use thread dim for shared-memory/parallelization
#ifndef ALLOW_ATOMIC_REDUCTION
    CUDA_LONG reductionBlock = blockIdx.z;                         // reduction-block index  --larger reductions are split into blocks
    pointers[pointers.size() - 1] += numElements * reductionBlock; // the output tensor is dense (no gaps); and there is one copy for each reduction block (those get further reduced into one later)
#endif
    if (id < numElements)                               // note: we have __syncthread() calls but only entire blocks in sync, so this is OK
        TensorOpElement<NUM_ARGS, REDUCTION_RANK, REGULAR_RANK, true, REGULAR_RANK - 1>::Compute(
            id, beta, pointers,
            alpha, op, reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize,
            regularOpStrideDivmod, reducingOpDimDivmod);
}

// helper function to provide a reduction buffer
template <class ElemType>
static shared_ptr<ElemType> AllocateReductionBuffer(size_t NUM_ARGS)
{
    ElemType* deviceBufferPtr;
    CUDA_CALL(hipMalloc((void**)&deviceBufferPtr, sizeof(ElemType) * NUM_ARGS));
    return shared_ptr<ElemType>(deviceBufferPtr, [](ElemType* deviceBufferPtr){ hipFree((void*)deviceBufferPtr); });
}

template <class ElemType>
static shared_ptr<ElemType> GetReductionBuffer(size_t NUM_ARGS)
{
    bool dontCache = false;         // (for debugging only)
    if (t_stream != 0 || dontCache) // we cache for the NULL stream but don't bother for others, since we only ever use the NULL stream currently
        return AllocateReductionBuffer<ElemType>(NUM_ARGS);

    static shared_ptr<ElemType> reductionBuffersCache[32]; // cache of objects    --TODO: Do we have a #define the max somewhere? Then also use it in CPUMatrix.cu GetOnesTensor()
    static size_t reductionBuffersCacheSize[_countof(reductionBuffersCache)] = { 0 };
    let deviceId = GridDim::GetCurrentDeviceId();
    if (deviceId >= _countof(reductionBuffersCache)) // index check w.r.t. our hard-coded dimensions
        return AllocateReductionBuffer<ElemType>(NUM_ARGS); // out of bounds: don't cache

    static std::once_flag initializedFlag[_countof(reductionBuffersCache)];
    std::call_once(initializedFlag[deviceId], [deviceId, NUM_ARGS]
    {
        reductionBuffersCache[deviceId] = AllocateReductionBuffer<ElemType>(NUM_ARGS);
        reductionBuffersCacheSize[deviceId] = NUM_ARGS;
    });

    if (NUM_ARGS > reductionBuffersCacheSize[deviceId]) // buffer size check
        LogicError("GetReductionBuffer: Must be called with the number of multiprocs, which may not change.");
    return reductionBuffersCache[deviceId];
}

// All dimensions (NUM_ARGS-ariness, number of input dimensions REGULAR_RANK and number of reduction dimensions REDUCTION_RANK) are bound to template parameters now.
template <class ElemType, C_size_t NUM_ARGS, C_int REDUCTION_RANK, C_int REGULAR_RANK>
static void LaunchTensorOpWithReduction(ElemType beta, array<ElemType*, NUM_ARGS> pointerVector,
                                        ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                        const SmallVector<size_t>& regularOpDims,       const array<SmallVector<ptrdiff_t>, NUM_ARGS>& regularStrideVectors,
                                        const SmallVector<size_t>& reducingOpDimVector, const array<SmallVector<ptrdiff_t>, NUM_ARGS>& reducingStrideVectors)
{
    // copy all parameters to CUDA-compatible data structures
    FixedArray<ElemType*, NUM_ARGS> pointers(pointerVector);
    SmallVector<C_size_t> regularOpStrideVector; // kernel needs the strides for converting thread index back to multi-dimensional tensor index
    C_size_t numElements = 1;
    // input divisors
    SmallVector<fast_divmod> regularOpStrideDivmodVector;
    for (C_size_t k = 0; k < regularOpDims.size(); k++)
    {
        regularOpStrideVector.push_back(numElements); // stride for dense representation of our output elements (if they were flattened)
        regularOpStrideDivmodVector.push_back(fast_divmod((unsigned int)numElements));
        numElements *= (C_size_t) regularOpDims[k];
    }
    // output divisors
    SmallVector<fast_divmod> reducingOpDimDivmodVector;
    C_size_t stride = 1;
    for (C_size_t k = 0; k < reducingOpDimVector.size(); ++k)
    {
        reducingOpDimDivmodVector.push_back(fast_divmod(stride));
        stride *= (C_size_t)reducingOpDimVector[k];
    }

    FixedArray<C_unsigned_int,           REGULAR_RANK>   regularOpStrides(regularOpStrideVector);
    FixedMatrix<C_int,         NUM_ARGS, REGULAR_RANK>   regularStrides(regularStrideVectors);
    FixedArray<C_unsigned_int,           REDUCTION_RANK> reducingOpDims(reducingOpDimVector);
    FixedMatrix<C_int,         NUM_ARGS, REDUCTION_RANK> reducingStrides(reducingStrideVectors);
    // reduced divisors
    FixedArray<fast_divmod,              REGULAR_RANK>   regularOpStrideDivmod(regularOpStrideDivmodVector);
    FixedArray<fast_divmod,              REDUCTION_RANK> reducingOpDimDivmod(reducingOpDimDivmodVector);

    // launch the kernel
    CUDA_LONG NN = (CUDA_LONG) numElements; // linear space identifying each individual output element
    SyncGuard syncGuard;

    // do some optimization for reductions
    //  - example: 30 GPU procs, warp size 32 --> 960 GPU cores
    //  - NN elements must be computed, each involving a reduction over reductionDim elements
    // Cases:
    //  - #output elements NN >= GPU cores  -->  use one proc per element, do reduction in inner loop
    //    E.g. if >=960 elements are computed, each gets its own GPU thread.
    //  - reduction dimension would benefit from multiple blocks  -->  multiple blocks work on a single output element
    //    E.g.
    //     - gradient of adding a bias: reducing to a bias, e.g. 512-dim
    //     - gradient of scalar multiplication: big elementwise product reduced to a scalar (big dot product, e.g. [1024 x 1024] = 1M elements)
    //     - softmax in seq-2-seq attention model: reduce over length of attention window (e.g. 20)
    //     - summation of criterion value: scalar reduction over a few hundred or thousand samples in the minibatch
    C_size_t reductionDim = 1; // number of elements to reduce over
    for (C_size_t k = 0; k < reducingOpDimVector.size(); k++)
        reductionDim *= (C_size_t) reducingOpDimVector[k];
    GridDim grid(NN);
    let& props = GridDim::GetDeviceProps();
    bool disableParallelReduction = false;                       // (for debugging)

    // === arg based reduction, one thread per output element
    if ((reductionOp == ElementWiseOperator::opArgmax) ||
        (reductionOp == ElementWiseOperator::opArgmin))
    {
        _launchTensorArgOp<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK> << <grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream >> > (
            pointers, reductionOp,
            regularOpStrides, regularStrides, grid.m_N,
            reducingOpDims, reducingStrides,
            regularOpStrideDivmod, reducingOpDimDivmod);
    }
    // === simple case: NN large, one thread per output element
    else if (reductionDim == 1 ||                                     // no reduction
             grid.m_blocksPerGrid >= props.multiProcessorCount ||     // enough output elements to fill all multiprocs
             reductionDim * numElements <= 2 * props.warpSize ||      // trivial operation not worth the trouble (2* because the more complex one also needs 2 kernel launches)
             disableParallelReduction ||                              // (for debugging)
             reductionDim * numElements <= props.multiProcessorCount) // recursive call from reduction below
    {
        // we got enough elements to generate: do one element per thread, and reduction inside
        _launchTensorOp<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(
            beta, pointers, alpha, op, reductionOp,
            regularOpStrides, regularStrides, grid.m_N,
            reducingOpDims, reducingStrides,
            regularOpStrideDivmod, reducingOpDimDivmod);
    }
    // === optimization: simple case would not use all multiprocs
    else
    {
        // m_blocksPerGrid can be thought of NN / 512, with appropriate rounding

        // we are reducing and are underutilizing the multiprocs we have: get more parallelism by doing reduction in parallel
        // If we get here, then
        //  - the total number of outputs to produce is < #multiprocs * warpSize, e.g. < 960
        //  - each output has at least two inputs, but possibly millions
        // Examples:
        //  (a1) NN=900
        //        - each multiproc processes multiple elements concurrently, each reducing over its inputs inside
        //        - use one block per output element
        //  (a2) NN=30
        //        - same as (a1) except 30 multiprocs run only a single block each
        //  (a3) NN=16
        //        - same as (a1) except only 16 multiproc run one block
        //  (b1) NN=15
        //        - 2 blocks work together on a single output element
        //  (b2) NN=1    (NN < #multiprocs, e.g. NN < 30)
        //        - multiple blocks work together on a single output element
        //        - only this case requires memory, and only K * NN
        //          where K = blocks that work together,
        //          both K and NN < #multiprocs,
        //          and K * NN = on the order of NN, but generally a bit larger due to rounding.

        // By how much do we underutilize?
        // We increase #blocks by that factor by breaking reduction into that many chunks.
        let numReductionChunks = max(props.multiProcessorCount / NN, 1); // only >1 for NN < multiProcessorCount

        // distribute NN over block X and Y
        let blockXOverBy = CeilDiv(NN, props.maxGridSize[0]);
        let numBlocksX = CeilDiv(NN, blockXOverBy);
        let numBlocksY = CeilDiv(NN, numBlocksX);
        // while block Z is for multiple blocks working together on a single output element
        let numBlocksZ = numReductionChunks;
        // Block dim is now:
        //  - X, Y: such that X*Y covers NN
        //  - Z: reduction chunks

        // reduction goes into thread dim X
        let reductionChunkSize = CeilDiv(reductionDim, numReductionChunks);
        let numThreadsX = min(reductionChunkSize, GridDim::maxThreadsPerBlock); // any that's over will be done by looping inside the kernel

        // --- cases (a1) and (a2)
        // This involves no reduction across blocks.
        if (numReductionChunks == 1)
        {
            _launchTensorOpWithReduction<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK><<<dim3(numBlocksX, numBlocksY, numBlocksZ), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(
                beta, pointers, alpha, op, reductionOp,
                regularOpStrides, regularStrides, NN,
                reducingOpDims, reducingStrides, /*reductionBegin*/ 0, reductionChunkSize,
                regularOpStrideDivmod, reducingOpDimDivmod);
        }
        // --- case (b)
        // Reduction across blocks. This is the difficult one.
#ifndef ALLOW_ATOMIC_REDUCTION // temporarily disabled to ensure it is not causing the non-reproducability
        else
        {
            // we get here if NN <= #multiprocs
            assert(NN <= props.multiProcessorCount && numBlocksX == NN && numBlocksY == 1);
            // dims are:
            //  - numBlocksZ = numReductionChunks = how many multiprocs work together to produce one output element
            //  - numBlocksX = NN = number of output elements
            //  - numThreadsX = reductionChunkSize clipped to 512; reductionChunkSize > 512 is handled by an inner for loop inside of the kernel

            // we need memory for block outputs of dimension [numBlocksX x numBlocksZ]
            //  - total elements = NN * Floor(#multiprocs / NN) = <= #multiprocs
            let reductionBufferSize = props.multiProcessorCount;
            assert(reductionBufferSize >= NN * numBlocksZ);
            shared_ptr<ElemType> reductionBuffer = GetReductionBuffer<ElemType>(reductionBufferSize);

            // 'pointers', 'regularOpStrides', and 'regularStrides' are set up to point to the target memory.
            // We need to reroute them to point to our reductionBuffer.
            //  - pointer[NUM_ARGS-1] -> replace by reductionBuffer
            //  - regularStrides -> replace [NUM_ARGS-1] by regularOpStrides which already represent the NN elements for a dense memory layout
            //  - beta -> 0 since we write into temp memory
            //  - kernel must use block.z as second index into the output buffer; add (block.z * NN) to the pointer
            FixedArray<ElemType*, NUM_ARGS> pointers1 = pointers;
            pointers1[NUM_ARGS - 1] = reductionBuffer.get();
            auto regularStrideVectors1 = regularStrideVectors;
            for (size_t k = 0; k < regularOpStrides.size(); k++)
                regularStrideVectors1[NUM_ARGS - 1][k] = (ptrdiff_t)regularOpStrideVector[k];
            FixedMatrix<C_int, NUM_ARGS, REGULAR_RANK> regularStrides1(regularStrideVectors1);
            ElemType beta1  = 0;
            ElemType alpha1 = 1;
            _launchTensorOpWithReduction<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK> << <dim3(numBlocksX, numBlocksY, numBlocksZ), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream >> >(
                beta1, pointers1, alpha1, op, reductionOp,
                regularOpStrides, regularStrides1, NN,
                reducingOpDims, reducingStrides, /*reductionBegin*/0, reductionChunkSize,
                regularOpStrideDivmod, reducingOpDimDivmod);

#if 1
            // now reduce and redistribute
            // Create a new tensor task, and execute it recursively:
            //  - input  = reductionBuffer
            //  - output = true output
            //  - op dims/strides     = output elements
            //  - reduce dims/strides = numBlocksZ
            //  - op = opCopy
            array<ElemType*, 2>                    pointerVector2{         reductionBuffer.get(),        pointerVector[NUM_ARGS - 1] };
            const array<SmallVector<ptrdiff_t>, 2> regularStrideVectors2{  regularStrideVectors1[NUM_ARGS - 1], regularStrideVectors[NUM_ARGS - 1] };
            const array<SmallVector<ptrdiff_t>, 2> reducingStrideVectors2{ SmallVector<ptrdiff_t>{ NN }, SmallVector<ptrdiff_t>{ 0 } };
            const SmallVector<size_t>              reducingOpDimVector2{ (size_t)numReductionChunks };
            LaunchTensorOpWithReduction<ElemType, /*NUM_ARGS=*/2, /*REDUCTION_RANK=*/1, REGULAR_RANK>(
                beta, pointerVector2, alpha, ElementWiseOperator::opCopy, reductionOp,
                regularOpDims, regularStrideVectors2,
                reducingOpDimVector2, reducingStrideVectors2);
            // (note: ^^this will have a nested syncGuard, which is fine)

#else
            _launchTensorOp<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(
                beta, pointers, alpha, op, reductionOp,
                regularOpStrides, regularStrides, grid.m_N,
                reducingOpDims, reducingStrides);
            //for (size_t z = 0; z < numBlocksZ; z++)
            //    _launchTensorOpWithReduction<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK><<<dim3(numBlocksX, numBlocksY, 1), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(z == 0 ? beta : 1, pointers, alpha, op,
            //    regularOpStrides, regularStrides, NN,
            //    reducingOpDims, reducingStrides, reductionChunkSize * z, reductionChunkSize);
            vector<ElemType> peekPartial(NN * numBlocksZ, -42);
            vector<ElemType> peekFinal(NN, -42);
            CUDA_CALL(hipMemcpy(peekPartial.data(), reductionBuffer,             sizeof(ElemType) * peekPartial.size(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(peekFinal.data(),   pointers[pointers.size()-1], sizeof(ElemType) * peekFinal.size(),   hipMemcpyDeviceToHost));
            double s1 = 0, s2 = 0;
            for (auto v : peekPartial)
                s1 += v;
            for (auto v : peekFinal)
                s2 += v;
            sin(1.0);
#endif
        }
#else
        else if (beta == 1)
        {
            // no need to pre-scale; just add (common for gradients)
            _launchTensorOpWithReduction<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK><<<dim3(numBlocksX, numBlocksY, numBlocksZ), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(beta, pointers, alpha, op, reductionOp, regularOpStrides,
                                                                   regularStrides, NN, reducingOpDims, reducingStrides, 0, reductionChunkSize,
                                                                   regularOpStrideDivmod, reducingOpDimDivmod);
            return;
        }
        else
        {
            // We need more than one chunk, we will use atomicAdd().
            // First reset/pre-multiply input; then do the remaining chunks using atomicAdd().
            _launchTensorOpWithReduction<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK><<<dim3(numBlocksX, numBlocksY, 1), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(beta, pointers, alpha, op, reductionOp, regularOpStrides, regularStrides, NN, reducingOpDims, reducingStrides, 0, reductionChunkSize,
                                                                   regularOpStrideDivmod, reducingOpDimDivmod);
            // We will leave it like this for a while, but eventually need to revisit using temporary memory.
            _launchTensorOpWithReduction<ElemType, NUM_ARGS, REDUCTION_RANK, REGULAR_RANK><<<dim3(numBlocksX, numBlocksY, numBlocksZ - 1), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(/*beta=*/1, pointers, alpha, op, reductionOp, regularOpStrides, regularStrides, NN, reducingOpDims, reducingStrides, reductionChunkSize, reductionChunkSize,
                                                                   regularOpStrideDivmod, reducingOpDimDivmod);
        }
#endif
    }
}

// -----------------------------------------------------------------------
// kernel and launch  --linear unary
// -----------------------------------------------------------------------

// for linear unary ops, we need to define a functor for every function for use as a template parameter (lambda syntax doesn't work in CUDA 7)
#define DefineUnaryTensorFunctor(oper)           \
    struct Functor##oper                         \
    {                                            \
        template <class ElemType>                \
        static __device__ ElemType f(ElemType a) \
        {                                        \
            return Op##oper(a);                  \
        }                                        \
    };
ForAllUnaryOps(DefineUnaryTensorFunctor);

// the top-level kernel for linear unary ops
// Note: If we have a beta, we have 2 memory accesses, so this optimization may no longer be needed as we are memory-bound.
template <class ElemType, class FN>
__global__ void _launchUnaryTensorOp(ElemType beta, const ElemType* pa, ElemType* pb, ElemType alpha, CUDA_LONG numElements)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id >= numElements)
        return;
    ElemType a = pa[id];
    ElemType val = FN::f(a);
    val *= alpha;
    if (beta != 0)
        val += beta * pb[id];
    pb[id] = val;
}
// version without beta and alpha
template <class ElemType, class FN>
__global__ void _launchUnaryTensorOp(const ElemType* pa, ElemType* pb, CUDA_LONG numElements)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id >= numElements)
        return;
    ElemType a = pa[id];
    ElemType val = FN::f(a);
    pb[id] = val;
}

// special case of linear unary operation
template <class ElemType>
void UnaryGPUTensorOp(ElemType beta, const ElemType* pa, ElemType* pb, ElemType alpha, ElementWiseOperator op, size_t regularOpDim)
{
    CUDA_LONG NN = (CUDA_LONG) regularOpDim;

#define CaseLaunchUnaryTensorOp(oper)                                                                                                        \
    case ElementWiseOperator::op##oper:                                                                                                      \
        if (beta == 0 && alpha == 1)                                                                                                         \
            _launchUnaryTensorOp<ElemType, Functor##oper><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(pa, pb, NN); \
        else                                                                                                                                 \
            _launchUnaryTensorOp<ElemType, Functor##oper><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(beta, pa, pb, alpha, NN);\
        break;

    SyncGuard syncGuard;
    GridDim grid(NN);
    switch (op)
    {
        ForAllUnaryOps(CaseLaunchUnaryTensorOp);
    default:
        LogicError("LaunchTensorOp1: Unknown op code %d.", (int) op);
    }
}

// -----------------------------------------------------------------------
// map runtime parameters NUM_ARGS to template parameters
// -----------------------------------------------------------------------

// tensor operation with k+1 dimensions (-1 means scalar)
template <class ElemType, C_size_t NUM_ARGS, C_int REGULAR_RANK>
static void TensorOpWithRegularLoop(ElemType beta, const array<ElemType*, NUM_ARGS>& pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                    const SmallVector<size_t>& regularOpDims,  const array<SmallVector<ptrdiff_t>, NUM_ARGS>& regularStrides,
                                    const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, NUM_ARGS>& reducingStrides)
{
    size_t reductionRank = reducingOpDims.size();
    switch (reductionRank)
    {
    case 2: return LaunchTensorOpWithReduction<ElemType, NUM_ARGS, /*REDUCTION_RANK=*/2, REGULAR_RANK>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 1: return LaunchTensorOpWithReduction<ElemType, NUM_ARGS, /*REDUCTION_RANK=*/1, REGULAR_RANK>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 0: return LaunchTensorOp             <ElemType, NUM_ARGS,                       REGULAR_RANK>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides);
    default:
        LogicError("TensorOp: %d non-flattened reduction dimensions are not supported.", (int)reductionRank);
    }
}

// tensor operation, generalized in number of arguments
// This function now expands into different k. It also eliminates the offsets by adding them to the pointers.
template <class ElemType, C_size_t NUM_ARGS>
void GPUTensorOp(ElemType beta, array<ElemType*, NUM_ARGS> pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
               const array<size_t, NUM_ARGS>& offsets,
               const SmallVector<size_t>& regularOpDims,  const array<SmallVector<ptrdiff_t>, NUM_ARGS>& regularStrides,
               const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, NUM_ARGS>& reducingStrides)
{
    for (C_size_t i = 0; i < NUM_ARGS; i++) // NUM_ARGS = a small constant, this will be unrolled
        pointers[i] += offsets[i];
    size_t regularRank = regularOpDims.size();
    switch (regularRank)
    {
    // N.B. consider code size impact when adding more cases.
    case 5: return TensorOpWithRegularLoop<ElemType, NUM_ARGS, /*REGULAR_RANK=*/5>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 4: return TensorOpWithRegularLoop<ElemType, NUM_ARGS, /*REGULAR_RANK=*/4>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 3: return TensorOpWithRegularLoop<ElemType, NUM_ARGS, /*REGULAR_RANK=*/3>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 2: return TensorOpWithRegularLoop<ElemType, NUM_ARGS, /*REGULAR_RANK=*/2>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 1: return TensorOpWithRegularLoop<ElemType, NUM_ARGS, /*REGULAR_RANK=*/1>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 0: return TensorOpWithRegularLoop<ElemType, NUM_ARGS, /*REGULAR_RANK=*/0>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    default: LogicError("TensorOp: %d non-flattened input dimensions are not supported.", (int) regularRank);
    }
}

//------------------------------------------------------------------------
// explicit instantiations--these are being called from GPUMatrix.cu
//------------------------------------------------------------------------

template void GPUTensorOp<float, /*NUM_ARGS=*/1>(float beta, array<float*, 1> pointers, float alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 1>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 1>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 1>& reducingStrides);
template void GPUTensorOp<float, /*NUM_ARGS=*/2>(float beta, array<float*, 2> pointers, float alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 2>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 2>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 2>& reducingStrides);
template void GPUTensorOp<float, /*NUM_ARGS=*/3>(float beta, array<float*, 3> pointers, float alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 3>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 3>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 3>& reducingStrides);
template void GPUTensorOp<float, /*NUM_ARGS=*/4>(float beta, array<float*, 4> pointers, float alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 4>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 4>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 4>& reducingStrides);
template void GPUTensorOp<float, /*NUM_ARGS=*/5>(float beta, array<float*, 5> pointers, float alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 5>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 5>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 5>& reducingStrides);
template void GPUTensorOp<double, /*NUM_ARGS=*/1>(double beta, array<double*, 1> pointers, double alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 1>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 1>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 1>& reducingStrides);
template void GPUTensorOp<double, /*NUM_ARGS=*/2>(double beta, array<double*, 2> pointers, double alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 2>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 2>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 2>& reducingStrides);
template void GPUTensorOp<double, /*NUM_ARGS=*/3>(double beta, array<double*, 3> pointers, double alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 3>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 3>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 3>& reducingStrides);
template void GPUTensorOp<double, /*NUM_ARGS=*/4>(double beta, array<double*, 4> pointers, double alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 4>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 4>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 4>& reducingStrides);
template void GPUTensorOp<double, /*NUM_ARGS=*/5>(double beta, array<double*, 5> pointers, double alpha, ElementWiseOperator op, ElementWiseOperator reductionOp, const array<size_t, 5>& offsets, const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 5>& regularStrides, const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 5>& reducingStrides);

template void UnaryGPUTensorOp(float beta, const float* pa, float* pb, float alpha, ElementWiseOperator op, size_t regularOpDim);
template void UnaryGPUTensorOp(double beta, const double* pa, double* pb, double alpha, ElementWiseOperator op, size_t regularOpDim);

}}}

#endif // CPUONLY
